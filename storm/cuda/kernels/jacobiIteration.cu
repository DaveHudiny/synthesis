#include "hip/hip_runtime.h"
#include "jacobiIteration.h"

#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#define FULL_WARP_MASK 0xffffffff
#define USE_CUSPARSE

template<typename T, bool Relative>
struct equalModuloPrecision : public thrust::binary_function<T,T,T>
{
__host__ __device__ T operator()(const T &x, const T &y) const
{
    if (Relative) {
		if (y == 0) {
			return ((x >= 0) ? (x) : (-x));
		}
		const T result = (x - y) / y;
		return ((result >= 0) ? (result) : (-result));
    } else {
        const T result = (x - y);
		return ((result >= 0) ? (result) : (-result));
    }
}
};

/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param b 
 * @param D 
 * @param x 
 * @param n 
 */
template<typename ValueType>
__global__ void 
gather(const ValueType *b, const ValueType *D, ValueType *x, const uint_fast64_t n) {
    uint_fast64_t i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < n) {
        x[i] = (b[i] - x[i]) * D[i];
    }
}

/**
 * @brief 
 * 
 * @param x 
 */
__inline__ __device__ 
uint_fast64_t prev_power_of_2 (uint_fast64_t x) {
    
    x = x | (x >> 1);
    x = x | (x >> 2);
    x = x | (x >> 4);
    x = x | (x >> 8);
    x = x | (x >> 16);
    x = x | (x >> 32);
    
    return x - (x >> 1);
}

/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param val 
 */
template<typename ValueType>
__inline__ __device__ 
ValueType warp_reduce (ValueType val) {

    val += __shfl_down_sync(FULL_WARP_MASK, val, 16);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 8);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 4);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 2);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 1);

    return val;
}

/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param col_ids 
 * @param row_ptr 
 * @param row_blocks 
 * @param data 
 * @param x 
 * @param y 
 */
template<typename ValueType>
__global__ void csr_spmv_adaptive_kernel (const uint_fast64_t *col_ids, const uint_fast64_t *row_ptr, const uint_fast64_t *row_blocks, const ValueType *data, const ValueType *x, ValueType *y) {

    const uint_fast64_t block_row_begin = row_blocks[blockIdx.x];
    const uint_fast64_t block_row_end = row_blocks[blockIdx.x + 1];
    const uint_fast64_t nnz_per_block = row_ptr[block_row_end] - row_ptr[block_row_begin]; 

    __shared__ ValueType cache[NNZ_PER_WG];

    if (block_row_end - block_row_begin > 1) {
        // CSR-Stream case...
        const uint_fast64_t i = threadIdx.x;
        const uint_fast64_t block_data_begin = row_ptr[block_row_begin];
        const uint_fast64_t thread_data_begin = block_data_begin + i;

        /**
         * Some block of rows may contain less than NNZ_PER_WG elements. But at 
         * most NNZ_PER_WG elements. (NNZ_PER_WG == blockDim.x -- Each thread has
         * one non-zero element to process.)
         */

        if (i < nnz_per_block) 
            cache[i] = data[thread_data_begin] * x[col_ids[thread_data_begin]];
        __syncthreads ();
        
        /**
         * If matrix is too sparse each thread will reduce 1 row. So one row
         * is reduced by one thread. In case that matrix is not too sparse and there
         * is more non-zero elements in one row than one row is reduced by multiple 
         * threads.
         */
        const uint_fast64_t threads_for_reduction = prev_power_of_2(blockDim.x/(block_row_end - block_row_begin));
            
        if (threads_for_reduction > 1) {
            // Reduce all non zeroes of row by multiple thread
            const uint_fast64_t thread_in_block = i % threads_for_reduction;
            const uint_fast64_t local_row = block_row_begin + i/threads_for_reduction;

            ValueType sum = 0.0;

            if (local_row < block_row_end) {
                const uint_fast64_t local_first_element = row_ptr[local_row] - row_ptr[block_row_begin];
                const uint_fast64_t local_last_element = row_ptr[local_row + 1] - row_ptr[block_row_begin];

                for(uint_fast64_t local_element = local_first_element + thread_in_block;
                    local_element < local_last_element;
                    local_element += threads_for_reduction) 
                    {
                        sum += cache[local_element];    
                    }
            }
            __syncthreads();
            cache[i] = sum;

            // Now each row has "threads_for_reduction" values in cache
            for (int j = threads_for_reduction / 2; j > 0; j >>= 1) {
                // Reduce for each row
                __syncthreads();

                const bool use_result = thread_in_block < j && i + j < NNZ_PER_WG;

                if (use_result)
                    sum += cache[i+j];
                __syncthreads();

                if(use_result)
                    cache[i] = sum;
            }
          
            if(thread_in_block == 0 && local_row < block_row_end)
                y[local_row] = sum;
        
        } else {
            // Reduce all non zeros of row by single thread
            uint_fast64_t local_row = block_row_begin + i;

            while (local_row < block_row_end) {
                ValueType sum = 0.0;

                for (uint_fast64_t j = row_ptr[local_row] - block_data_begin;
                    j < row_ptr[local_row + 1] - block_data_begin;
                    j++) 
                    {
                        sum += cache[j];
                    }
                y[local_row] = sum;
                local_row += NNZ_PER_WG;
            }
        }
    } else {
        /**
         * Row Block == Row => NNZ_PER_WG <= nnz_per_block
         * If a block of rows contains only one row, then it can have 
         * NNZ_PER_WG (blockDim.x) <= nnz_per_block (also blockDim.x > nnz_per_block 
         * is possible). 
         */
        const uint_fast64_t nnz_per_block = row_ptr[block_row_end] - row_ptr[block_row_begin]; 

        if (nnz_per_block <= 64) {
            // CSR-Vector case...
            const uint_fast64_t thread_id = threadIdx.x;
            const uint_fast64_t warp_id = thread_id / warpSize;
            ValueType sum = 0.0;

            if (warp_id == 0) {
                // only one warp processes a whole row
                const uint_fast64_t row_start = row_ptr[block_row_begin];
                const uint_fast64_t row_end = row_ptr[block_row_end];

                for(uint_fast64_t element = row_start + thread_id; element < row_end; element += warpSize) {
                    sum += data[element] * x[col_ids[element]];
                }

                sum = warp_reduce<ValueType>(sum);

                if(thread_id == 0)
                    y[block_row_begin] = sum;
            }
        }  else {
            // CSR-VectorL case...
            const uint_fast64_t thread_id = threadIdx.x;
             
            const uint_fast64_t row_start = row_ptr[block_row_begin];
            const uint_fast64_t row_end = row_ptr[block_row_end];
            
            ValueType sum = 0.0;
            for(uint_fast64_t element = row_start + thread_id; element < row_end; element += blockDim.x)
                sum += data[element] * x[col_ids[element]];

            cache[thread_id] = sum;
            __syncthreads();

            for(int stride = blockDim.x / 2; stride > 0;  stride >>= 1) {
                if(thread_id < stride) 
                    cache[thread_id] += cache[thread_id + stride];
                __syncthreads();
            }

            __syncthreads();

            if(thread_id == 0) 
                y[block_row_begin] = cache[thread_id];
        }
    }
}

template<bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
bool jacobiIteration_solver(
                uint_fast64_t const maxIterationCount, 
                ValueType const precision,
                uint_fast64_t const matrixRowCount,
                uint_fast64_t const matrixNnzCount,
                uint_fast64_t const matrixBlockCount,
                std::vector<ValueType>& x,
                std::vector<ValueType> const& b,
                std::vector<ValueType> const& nnzValues,
                std::vector<ValueType> const& D,
                std::vector<uint_fast64_t> const& columnIndices,
                std::vector<uint_fast64_t> const& rowStartIndices,
                std::vector<uint_fast64_t> const& rowBlocks,
                size_t& iterationCount)
{
    std::cout << "CUDA Jacobi method\n";

    uint_fast64_t* device_columnIndices = nullptr;
    uint_fast64_t* device_rowStartIndices = nullptr; 
    uint_fast64_t* device_rowBlocks = nullptr;

    ValueType* device_x = nullptr;
    ValueType* device_xSwap = nullptr;
    ValueType* device_b = nullptr;
    ValueType* device_D = nullptr;
    ValueType* device_nnzValues = nullptr;

    bool converged = false;
    iterationCount = 0;

    // Device memory allocation
    hipError_t cudaMallocResult;
    bool errorOccured = false;

    dim3 gatherGridDim(ceil(double(matrixRowCount)/NNZ_PER_WG));

#ifdef USE_CUSPARSE
    ValueType            alpha      = 1.0f;
    ValueType            beta       = 0.0f;
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
#endif

    cudaMallocResult = hipMalloc<uint_fast64_t>((&device_columnIndices), sizeof(uint_fast64_t) * columnIndices.size());
    if (cudaMallocResult != hipSuccess) {
        std::cout << "Could not allocate memory for Matrix Column Indices, Error Code " << cudaMallocResult << "." << std::endl;
        errorOccured = true;
        goto cleanup;
    }

    cudaMallocResult = hipMalloc<uint_fast64_t>((&device_rowStartIndices), sizeof(uint_fast64_t) * (matrixRowCount + 1));
    if (cudaMallocResult != hipSuccess) {
        std::cout << "Could not allocate memory for Row Start Indices, Error Code " << cudaMallocResult << "." << std::endl;
        errorOccured = true;
        goto cleanup;
    }

#ifndef USE_CUSPARSE
    cudaMallocResult = hipMalloc<uint_fast64_t>((&device_rowBlocks), sizeof(uint_fast64_t) * (matrixBlockCount + 1));
    if (cudaMallocResult != hipSuccess) {
        std::cout << "Could not allocate memory for Row Blocks, Error Code " << cudaMallocResult << "." << std::endl;
        errorOccured = true;
        goto cleanup;
    }
#endif

    cudaMallocResult = hipMalloc<ValueType>((&device_x), sizeof(ValueType) * matrixRowCount);
    if (cudaMallocResult != hipSuccess) {
        std::cout << "Could not allocate memory for Vector x, Error Code " << cudaMallocResult << "." << std::endl;
        errorOccured = true;
        goto cleanup;
    }

    cudaMallocResult = hipMalloc<ValueType>((&device_xSwap), sizeof(ValueType) * matrixRowCount);
    if (cudaMallocResult != hipSuccess) {
        std::cout << "Could not allocate memory for Vector x swap, Error Code " << cudaMallocResult << "." << std::endl;
        errorOccured = true;
        goto cleanup;
    }

    cudaMallocResult = hipMalloc<ValueType>((&device_b), sizeof(ValueType) * matrixRowCount);
    if (cudaMallocResult != hipSuccess) {
        std::cout << "Could not allocate memory for Vector b, Error Code " << cudaMallocResult << "." << std::endl;
        errorOccured = true;
        goto cleanup;
    }

    cudaMallocResult = hipMalloc<ValueType>((&device_D), sizeof(ValueType) * matrixRowCount);
    if (cudaMallocResult != hipSuccess) {
        std::cout << "Could not allocate memory for Vector D, Error Code " << cudaMallocResult << "." << std::endl;
        errorOccured = true;
        goto cleanup;
    }

    cudaMallocResult = hipMalloc<ValueType>((&device_nnzValues), sizeof(ValueType) * matrixNnzCount);
    if (cudaMallocResult != hipSuccess) {
        std::cout << "Could not allocate memory for Vector of Matrix Values, Error Code " << cudaMallocResult << "." << std::endl;
        errorOccured = true;
        goto cleanup;
    }
    // Memory allocated, copy data to device
    hipError_t cudaCopyResult;

    cudaCopyResult = hipMemcpy(device_columnIndices, columnIndices.data(), sizeof(uint_fast64_t) * columnIndices.size(), hipMemcpyHostToDevice);
    if (cudaCopyResult != hipSuccess) {
        std::cout << "Could not copy data for Matrix Column Indices, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
    }

    cudaCopyResult = hipMemcpy(device_rowStartIndices, rowStartIndices.data(), sizeof(uint_fast64_t) * (matrixRowCount + 1), hipMemcpyHostToDevice);
    if (cudaCopyResult != hipSuccess) {
        std::cout << "Could not copy data for Matrix Row Start Indices, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
    }

#ifndef USE_CUSPARSE
    cudaCopyResult = hipMemcpy(device_rowBlocks, rowBlocks.data(), sizeof(uint_fast64_t) * (matrixBlockCount + 1), hipMemcpyHostToDevice);
    if (cudaCopyResult != hipSuccess) {
        std::cout << "Could not copy data for Matrix Row Blocks, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
    }
#endif
    
    cudaCopyResult = hipMemcpy(device_x, x.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice);
    if (cudaCopyResult != hipSuccess) {
        std::cout << "Could not copy data for Vector x, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
    }

    cudaCopyResult = hipMemset(device_xSwap, 0, sizeof(ValueType) * matrixRowCount);
    if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not zero the Swap Vector x, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
    }

    cudaCopyResult = hipMemcpy(device_b, b.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice);
    if (cudaCopyResult != hipSuccess) {
        std::cout << "Could not copy data for Vector b, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
    }

    cudaCopyResult = hipMemcpy(device_D, D.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice);
    if (cudaCopyResult != hipSuccess) {
        std::cout << "Could not copy data for Vector D, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
    }
    
    cudaCopyResult = hipMemcpy(device_nnzValues, nnzValues.data(), sizeof(ValueType) * matrixNnzCount, hipMemcpyHostToDevice);
    if (cudaCopyResult != hipSuccess) {
        std::cout << "Could not copy data for Matrix, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
    }

#ifdef USE_CUSPARSE
    // CUSPARSE APIs
    hipsparseCreate(&handle); 
    // Create sparse matrix A in CSR format
    hipsparseCreateCsr(&matA, matrixRowCount, matrixRowCount, matrixNnzCount, (void*)device_rowStartIndices, (void*)device_columnIndices, (void*)device_nnzValues, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, CUDA_DATATYPE);
    // Create dense vector X
    hipsparseCreateDnVec(&vecX, matrixRowCount, (void*)device_x, CUDA_DATATYPE);
    // Create dense vector Y
    hipsparseCreateDnVec(&vecY, matrixRowCount, (void*)device_xSwap, CUDA_DATATYPE);
    // allocate an external buffer
    hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE,
        HIPSPARSE_CSRMV_ALG2, &bufferSize); 
    hipMalloc(&dBuffer, bufferSize); 
#endif

    // Data is on device, start Kernel
    while (!converged && iterationCount < maxIterationCount) {
        // call kernels 
#ifdef USE_CUSPARSE        
        hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, dBuffer);
#else
        csr_spmv_adaptive_kernel<ValueType><<<matrixBlockCount, NNZ_PER_WG>>>(device_columnIndices, device_rowStartIndices, device_rowBlocks, device_nnzValues, device_x, device_xSwap); 
#endif
        gather<ValueType><<<gatherGridDim, NNZ_PER_WG>>>(device_b, device_D, device_xSwap, matrixRowCount);

        // Check for convergence
		// Transform: x = abs(x - xSwap)/ xSwap
		thrust::device_ptr<ValueType> devicePtrThrust_x(device_x);
		thrust::device_ptr<ValueType> devicePtrThrust_x_end(device_x + matrixRowCount);
        thrust::device_ptr<ValueType> devicePtrThrust_xSwap(device_xSwap);
        thrust::transform(devicePtrThrust_x, devicePtrThrust_x_end, devicePtrThrust_xSwap, devicePtrThrust_x, equalModuloPrecision<ValueType, Relative>());
        // Reduce: get Max over x and check for res < Precision
		ValueType maxX = thrust::reduce(devicePtrThrust_x, devicePtrThrust_x_end, -std::numeric_limits<ValueType>::max(), thrust::maximum<ValueType>());
		converged = (maxX < precision);
		++iterationCount;

		// Swap pointers, device_x always contains the most current result
        std::swap(device_x, device_xSwap);

#ifdef USE_CUSPARSE
        // set new values for vecX and vecY
        hipsparseDnVecSetValues(vecX, (void*)device_x);
        hipsparseDnVecSetValues(vecY, (void*)device_xSwap);
#endif
    }

    if (!converged && (iterationCount == maxIterationCount)) {
		iterationCount = 0;
		errorOccured = true;
    }
    
    // Get x (result) back from the device
	cudaCopyResult = hipMemcpy(x.data(), device_x, sizeof(ValueType) * matrixRowCount, hipMemcpyDeviceToHost);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy back data for result vector x, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
	}

    // All code related to freeing memory and clearing up the device
cleanup:
    hipError_t cudaFreeResult;

    if (device_columnIndices != nullptr) {
        cudaFreeResult = hipFree(device_columnIndices);
        if (cudaFreeResult != hipSuccess) {
            std::cout << "Could not free Memory of Matrix Column Indices, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
        }
        device_columnIndices = nullptr;
    }
    if (device_rowStartIndices != nullptr) {
        cudaFreeResult = hipFree(device_rowStartIndices);
        if (cudaFreeResult != hipSuccess) {
            std::cout << "Could not free Memory of Row Start Indices, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
        }
        device_rowStartIndices = nullptr;
    }
    if (device_rowBlocks != nullptr) {
        cudaFreeResult = hipFree(device_rowBlocks);
        if (cudaFreeResult != hipSuccess) {
            std::cout << "Could not free Memory of Row Blocks, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
        }
        device_rowBlocks = nullptr;
    }
    if (device_x != nullptr) {
        cudaFreeResult = hipFree(device_x);
        if (cudaFreeResult != hipSuccess) {
            std::cout << "Could not free Memory of Vector x, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
        }
        device_x = nullptr;
    }
    if (device_xSwap != nullptr) {
        cudaFreeResult = hipFree(device_xSwap);
        if (cudaFreeResult != hipSuccess) {
            std::cout << "Could not free Memory of Vector x swap, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
        }
        device_xSwap = nullptr;
    }
    if (device_b != nullptr) {
        cudaFreeResult = hipFree(device_b);
        if (cudaFreeResult != hipSuccess) {
            std::cout << "Could not free Memory of Vector b, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
        }
        device_b = nullptr;
    }
    if (device_D != nullptr) {
        cudaFreeResult = hipFree(device_D);
        if (cudaFreeResult != hipSuccess) {
            std::cout << "Could not free Memory of Vector D, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
        }
        device_D = nullptr;
    }
    if (device_nnzValues != nullptr) {
        cudaFreeResult = hipFree(device_nnzValues);
        if (cudaFreeResult != hipSuccess) {
            std::cout << "Could not free Memory of Matrix Values, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
        }
        device_nnzValues = nullptr;
    }

#ifdef USE_CUSPARSE
    // destroy matrix/vector descriptors
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);
    
    // device memory deallocation
    hipFree(dBuffer);
#endif

    std::cout << "--------------------------------------------------------------\n";
    std::cout << "METRICS\n";
    std::cout << "Jacobi Iterations: " << iterationCount << "\n";
    std::cout << "Matrix dimension: " << matrixRowCount << "\n";
    std::cout << "Matrix nnz count: " << matrixNnzCount << "\n";
    std::cout << "--------------------------------------------------------------\n";

    return !errorOccured;
}

bool jacobiIteration_solver_double(uint_fast64_t const maxIterationCount, 
                double const precision,
                uint_fast64_t const matrixRowCount,
                uint_fast64_t const matrixNnzCount,
                uint_fast64_t const matrixBlockCount,
                std::vector<double> & x,
                std::vector<double> const& b,
                std::vector<double> const& nnzValues,
                std::vector<double> const& D,
                std::vector<uint_fast64_t> const& columnIndices,
                std::vector<uint_fast64_t> const& rowStartIndices,
                std::vector<uint_fast64_t> const& rowBlocks,
                size_t& iterationCount,
                bool const relativePrecisionCheck) {
    if (relativePrecisionCheck) {
        return jacobiIteration_solver<true, double, HIP_R_64F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    } else {
        return jacobiIteration_solver<false, double, HIP_R_64F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    }
}

bool jacobiIteration_solver_float(uint_fast64_t const maxIterationCount, 
                float const precision,
                uint_fast64_t const matrixRowCount,
                uint_fast64_t const matrixNnzCount,
                uint_fast64_t const matrixBlockCount,
                std::vector<float> & x,
                std::vector<float> const& b,
                std::vector<float> const& nnzValues,
                std::vector<float> const& D,
                std::vector<uint_fast64_t> const& columnIndices,
                std::vector<uint_fast64_t> const& rowStartIndices,
                std::vector<uint_fast64_t> const& rowBlocks,
                size_t& iterationCount,
                bool const relativePrecisionCheck) {
    if (relativePrecisionCheck) {
        return jacobiIteration_solver<true, float, HIP_R_32F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    } else {
        return jacobiIteration_solver<false, float, HIP_R_32F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    }
}
