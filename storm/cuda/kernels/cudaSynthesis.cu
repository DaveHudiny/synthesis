#include "hip/hip_runtime.h"
#include "cudaSynthesis.h"

#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/remove.h>
#include <thrust/replace.h>
#include <thrust/scan.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>

#include <cub/device/device_segmented_reduce.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#define FULL_WARP_MASK 0xffffffff
#define USE_CUSPARSE

#define CHECK_CUDA(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA API assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define CHECK_CUSPARSE(func) { cusparseAssert((func), __FILE__, __LINE__); }                                                  
inline void cusparseAssert(hipsparseStatus_t code, const char *file, int line, bool abort=true)
{                                                                             
    if (code != HIPSPARSE_STATUS_SUCCESS) {                                   
        fprintf(stderr,"CUSPARSE API assert: %s %s %d\n", hipsparseGetErrorString(code), file, line);    
        if (abort) exit(code);                                             
    }                                                                          
}

/*******************************************************************************/
/*                             DEVICE CODE                                     */
/*******************************************************************************/

template<typename T, bool Relative>
struct equalModuloPrecision : public thrust::binary_function<T,T,T>
{
__host__ __device__ T operator()(const T &x, const T &y) const
{
    if (Relative) {
		if (y == 0) {
			return ((x >= 0) ? (x) : (-x));
		}
		const T result = (x - y) / y;
		return ((result >= 0) ? (result) : (-result));
    } else {
        const T result = (x - y);
		return ((result >= 0) ? (result) : (-result));
    }
}
};

template<typename T>
struct is_minus_one
{
  __host__ __device__
  bool operator()(const T x) {
    return (x == T(-1.0));
  }
};

/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param b 
 * @param D 
 * @param x 
 * @param n 
 */
template<typename ValueType>
__global__ void 
gather(const ValueType *b, const ValueType *D, ValueType *x, const uint_fast64_t n) {
    uint_fast64_t i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < n) {
        x[i] = (b[i] - x[i]) * D[i];
    }
}

/**
 * @brief 
 * 
 * @param x 
 */
__inline__ __device__ 
uint_fast64_t prev_power_of_2 (uint_fast64_t x) {
    
    x = x | (x >> 1);
    x = x | (x >> 2);
    x = x | (x >> 4);
    x = x | (x >> 8);
    x = x | (x >> 16);
    x = x | (x >> 32);
    
    return x - (x >> 1);
}

/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param val 
 */
template<typename ValueType>
__inline__ __device__ 
ValueType sumSingleWarpReg (ValueType val) {

    val += __shfl_down_sync(FULL_WARP_MASK, val, 16);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 8);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 4);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 2);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 1);

    return val;
}


/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param val 
 * @param threads_per_row 
 */
 template<typename ValueType>
 __inline__ __device__ 
 ValueType minSingleWarpReg (ValueType val, ValueType minMaxInitializer, uint32_t threads_per_row) {
    ValueType localMin = minMaxInitializer;
    
    switch (threads_per_row) {
        case 32:
            localMin = __shfl_down_sync(FULL_WARP_MASK, val, 16); 
            val = (val < localMin) ? val : localMin;
        case 16:
            localMin = __shfl_down_sync(FULL_WARP_MASK, val, 8); 
            val = (val < localMin) ? val : localMin;
        case 8:
            localMin = __shfl_down_sync(FULL_WARP_MASK, val, 4); 
            val = (val < localMin) ? val : localMin;
        case 4:
            localMin = __shfl_down_sync(FULL_WARP_MASK, val, 2); 
            val = (val < localMin) ? val : localMin;
        case 2:
            localMin = __shfl_down_sync(FULL_WARP_MASK, val, 1); 
            val = (val < localMin) ? val : localMin;
    }

    return val;
 }

 /**
 * @brief 
 * 
 * @tparam ValueType 
 * @param val 
 * @param threads_per_row  
 */
 template<typename ValueType>
 __inline__ __device__ 
 ValueType maxSingleWarpReg (ValueType val, ValueType minMaxInitializer, uint32_t threads_per_row) {
    ValueType localMax = minMaxInitializer;
    
    switch (threads_per_row) {
        case 32:
            localMax = __shfl_down_sync(FULL_WARP_MASK, val, 16); 
            val = (val > localMax) ? val : localMax;
        case 16:
            localMax = __shfl_down_sync(FULL_WARP_MASK, val, 8); 
            val = (val > localMax) ? val : localMax;
        case 8:
            localMax = __shfl_down_sync(FULL_WARP_MASK, val, 4); 
            val = (val > localMax) ? val : localMax;
        case 4:
            localMax = __shfl_down_sync(FULL_WARP_MASK, val, 2); 
            val = (val > localMax) ? val : localMax;
        case 2:
            localMax = __shfl_down_sync(FULL_WARP_MASK, val, 1); 
            val = (val > localMax) ? val : localMax;
    }

    return val;
 }

/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param col_ids 
 * @param row_ptr 
 * @param row_blocks 
 * @param data 
 * @param x 
 * @param y 
 */
template<typename ValueType>
__global__ void csr_spmv_adaptable_kernel (const uint_fast64_t *col_ids, const uint_fast64_t *row_ptr, const uint_fast64_t *row_blocks, const ValueType *data, const ValueType *x, ValueType *y) {

    const uint_fast64_t block_begin = row_blocks[blockIdx.x];
    const uint_fast64_t block_end = row_blocks[blockIdx.x + 1];
    const uint_fast64_t nnz_per_block = row_ptr[block_end] - row_ptr[block_begin]; 

    __shared__ ValueType multRes[NNZ_PER_WG];

    if (block_end - block_begin > 1) {
        // CSR-Stream case...
        const uint_fast64_t thread_id = threadIdx.x;
        const uint_fast64_t block_data_begin = row_ptr[block_begin];
        const uint_fast64_t thread_data_begin = block_data_begin + thread_id;
        /**
         * Block of rows may contain less than NNZ_PER_WG elements. But at 
         * most NNZ_PER_WG elements. (NNZ_PER_WG == blockDim.x -- Each thread has
         * one non-zero element to process.)
         */
        if (thread_id < nnz_per_block) 
            multRes[thread_id] = data[thread_data_begin] * x[col_ids[thread_data_begin]];
        __syncthreads ();
        
        /**
         * If matrix is too sparse each thread will reduce 1 row. So one row
         * is reduced by one thread. In case that matrix is not too sparse and there
         * is more non-zero elements in one row than one row is reduced by multiple 
         * threads.
         */
        const uint_fast64_t threads_for_reduction = prev_power_of_2(blockDim.x/(block_end - block_begin));
        if (threads_for_reduction > 1) {
            // Reduce all non zeroes of row by multiple thread
            const uint_fast64_t thread_in_block = thread_id % threads_for_reduction;
            const uint_fast64_t local_row = block_begin + thread_id/threads_for_reduction;

            ValueType sum = 0.0;

            if (local_row < block_end) {
                const uint_fast64_t local_first_element = row_ptr[local_row] - row_ptr[block_begin];
                const uint_fast64_t local_last_element = row_ptr[local_row + 1] - row_ptr[block_begin];

                for(uint_fast64_t local_element = local_first_element + thread_in_block; local_element < local_last_element; local_element += threads_for_reduction) {
                    sum += multRes[local_element];    
                }
            }
            __syncthreads();
            multRes[thread_id] = sum;

            // Now each row has "threads_for_reduction" values in multRes
            for (int j = threads_for_reduction / 2; j > 0; j >>= 1) {
                // Reduce for each row
                __syncthreads();

                const bool use_result = thread_in_block < j && thread_id + j < NNZ_PER_WG;

                sum = (use_result) ? sum + multRes[thread_id + j] : sum;
                __syncthreads();

                if(use_result)
                    multRes[thread_id] = sum;
            }
          
            if(thread_in_block == 0 && local_row < block_end)
                y[local_row] = sum;
        
        } else {
            // Reduce all non zeros of row by single thread
            uint_fast64_t local_row = block_begin + thread_id;
            while (local_row < block_end) {
                ValueType sum = 0.0;

                for (uint_fast64_t j = row_ptr[local_row] - block_data_begin; j < row_ptr[local_row + 1] - block_data_begin; j++) {
                    sum += multRes[j];
                }
                y[local_row] = sum;
                local_row += NNZ_PER_WG;
            }
        }
    } else {
        /**
         * Row Block == Row => NNZ_PER_WG <= nnz_per_block
         * If a block of rows contains only one row, then it can have 
         * NNZ_PER_WG (blockDim.x) <= nnz_per_block (also blockDim.x > nnz_per_block 
         * is possible). 
         */
        const uint_fast64_t nnz_per_block = row_ptr[block_end] - row_ptr[block_begin]; 

        if (nnz_per_block <= 64) {
            // CSR-Vector case...
            const uint_fast64_t thread_id = threadIdx.x;
            const uint_fast64_t warp_id = thread_id / warpSize;
            ValueType sum = 0.0;

            if (warp_id == 0) {
                // only one warp processes a whole row
                const uint_fast64_t row_start = row_ptr[block_begin];
                const uint_fast64_t row_end = row_ptr[block_end];

                for(uint_fast64_t element = row_start + thread_id; element < row_end; element += warpSize) {
                    sum += data[element] * x[col_ids[element]];
                }

                sum = sumSingleWarpReg<ValueType>(sum);

                if(thread_id == 0)
                    y[block_begin] = sum;
            }
        }  else {
            // CSR-VectorL case...
            const uint_fast64_t thread_id = threadIdx.x;
             
            const uint_fast64_t row_start = row_ptr[block_begin];
            const uint_fast64_t row_end = row_ptr[block_end];
            
            ValueType sum = 0.0;
            for(uint_fast64_t element = row_start + thread_id; element < row_end; element += blockDim.x)
                sum += data[element] * x[col_ids[element]];

            multRes[thread_id] = sum;
            __syncthreads();

            for(int stride = blockDim.x / 2; stride > 0;  stride >>= 1) {
                if(thread_id < stride) 
                    multRes[thread_id] += multRes[thread_id + stride];
                __syncthreads();
            }

            __syncthreads();

            if(thread_id == 0) 
                y[block_begin] = multRes[thread_id];
        }
    }
}

template <typename ValueType, unsigned int GROUPS_PER_BLOCK, unsigned int THREADS_PER_GROUP, bool Maximize>
__launch_bounds__(GROUPS_PER_BLOCK * THREADS_PER_GROUP,1) 
__global__ void
segmented_vector_reduce_kernel(const uint_fast64_t num_groups, const uint_fast64_t * __restrict__ nondeterministicChoiceIndices, ValueType * __restrict__ result, const ValueType * __restrict__ input, const ValueType minMaxInitializer)
{
    __shared__ volatile uint_fast64_t groups_pointers[GROUPS_PER_BLOCK][2];
    
    const uint_fast64_t THREADS_PER_BLOCK = GROUPS_PER_BLOCK * THREADS_PER_GROUP;

    const uint_fast64_t thread_id   = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;// global thread index
    const uint_fast64_t thread_lane = threadIdx.x & (THREADS_PER_GROUP - 1);       // thread index within the group
    const uint_fast64_t group_id   = thread_id   /  THREADS_PER_GROUP;             // global vector index
    const uint_fast64_t group_lane = threadIdx.x /  THREADS_PER_GROUP;             // vector index within the block
    const uint_fast64_t num_states = GROUPS_PER_BLOCK * gridDim.x;                 // total number of active groups

    for(uint_fast64_t row = group_id; row < num_groups; row += num_states) {
        if(thread_lane < 2)
            groups_pointers[group_lane][thread_lane] = nondeterministicChoiceIndices[row + thread_lane];

        const uint_fast64_t group_start = groups_pointers[group_lane][0];
        const uint_fast64_t group_end   = groups_pointers[group_lane][1];

        // initialize local Min/Max
        ValueType localMinMax = minMaxInitializer;

        if (THREADS_PER_GROUP == 32 && group_end - group_start > 32) {
            // ensure aligned memory access
            uint_fast64_t index = group_start - (group_start & (THREADS_PER_GROUP - 1)) + thread_lane;

            if(index >= group_start && index < group_end) {
				if(Maximize) { localMinMax = (localMinMax < input[index]) ? input[index] : localMinMax; } 
                else         { localMinMax = (localMinMax > input[index]) ? input[index] : localMinMax; }
			}
            // accumulate local sums
            for(index += THREADS_PER_GROUP; index < group_end; index += THREADS_PER_GROUP) {
                if(Maximize) { localMinMax = (localMinMax < input[index]) ? input[index] : localMinMax; } 
                else         { localMinMax = (localMinMax > input[index]) ? input[index] : localMinMax; }
            }
        } else {
            // accumulate local sums
            for(uint_fast64_t index = group_start + thread_lane; index < group_end; index += THREADS_PER_GROUP)
                if(Maximize) { localMinMax = (localMinMax < input[index]) ? input[index] : localMinMax; } 
                else         { localMinMax = (localMinMax > input[index]) ? input[index] : localMinMax; }
        }

        // reduce local min/max to row min/max
        localMinMax = (Maximize) ? maxSingleWarpReg<ValueType>(localMinMax, minMaxInitializer, THREADS_PER_GROUP) : 
                                   minSingleWarpReg<ValueType>(localMinMax, minMaxInitializer, THREADS_PER_GROUP) ;

        if (thread_lane == 0)
            result[row] = (localMinMax == minMaxInitializer) ? 0.0 : localMinMax;
    }
}

/*******************************************************************************/
/*                               HOST CODE                                     */
/*******************************************************************************/

template<bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
bool jacobiIteration_solver(
                uint_fast64_t const maxIterationCount, 
                ValueType const precision,
                uint_fast64_t const matrixRowCount,
                uint_fast64_t const matrixNnzCount,
                uint_fast64_t const matrixBlockCount,
                std::vector<ValueType>& x,
                std::vector<ValueType> const& b,
                std::vector<ValueType> const& nnzValues,
                std::vector<ValueType> const& D,
                std::vector<uint_fast64_t> const& columnIndices,
                std::vector<uint_fast64_t> const& rowStartIndices,
                std::vector<uint_fast64_t> const& rowBlocks,
                size_t& iterationCount)
{

    bool errorOccured = false;
    bool converged = false;
    iterationCount = 0;

    uint_fast64_t* device_columnIndices = nullptr;
    uint_fast64_t* device_rowStartIndices = nullptr; 
    uint_fast64_t* device_rowBlocks = nullptr;

    ValueType* device_x = nullptr;
    ValueType* device_xSwap = nullptr;
    ValueType* device_b = nullptr;
    ValueType* device_D = nullptr;
    ValueType* device_nnzValues = nullptr;

    dim3 gatherGridDim(ceil(double(matrixRowCount)/NNZ_PER_WG));

#ifdef USE_CUSPARSE
    ValueType            alpha      = 1.0f;
    ValueType            beta       = 0.0f;
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
#endif

    // Device memory allocation
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_columnIndices), sizeof(uint_fast64_t) * columnIndices.size()) );
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_rowStartIndices), sizeof(uint_fast64_t) * (matrixRowCount + 1)) );
    
    CHECK_CUDA( hipMalloc<ValueType>((&device_x), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_xSwap), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_b), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_D), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_nnzValues), sizeof(ValueType) * matrixNnzCount) );
#ifndef USE_CUSPARSE
    CHECK_CUDA( cudaMallocResult = hipMalloc<uint_fast64_t>((&device_rowBlocks), sizeof(uint_fast64_t) * (matrixBlockCount + 1)) );
#endif

    // Memory allocated, copy data to device
    CHECK_CUDA( hipMemcpy(device_columnIndices, columnIndices.data(), sizeof(uint_fast64_t) * columnIndices.size(), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_rowStartIndices, rowStartIndices.data(), sizeof(uint_fast64_t) * (matrixRowCount + 1), hipMemcpyHostToDevice) );
    
    CHECK_CUDA( hipMemcpy(device_x, x.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(device_xSwap, 0, sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMemcpy(device_b, b.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_D, D.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_nnzValues, nnzValues.data(), sizeof(ValueType) * matrixNnzCount, hipMemcpyHostToDevice) );
#ifndef USE_CUSPARSE
    CHECK_CUDA( hipMemcpy(device_rowBlocks, rowBlocks.data(), sizeof(uint_fast64_t) * (matrixBlockCount + 1), hipMemcpyHostToDevice) );
#endif

    // CUSPARSE settings
#ifdef USE_CUSPARSE
    CHECK_CUSPARSE( hipsparseCreate(&handle) ); 
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, matrixRowCount, matrixRowCount, matrixNnzCount, (void*)device_rowStartIndices, (void*)device_columnIndices, (void*)device_nnzValues, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, matrixRowCount, (void*)device_x, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, matrixRowCount, (void*)device_xSwap, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, &bufferSize) ); 
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) ); 
#endif

    // Data is on device, start Kernel
    while (!converged && iterationCount < maxIterationCount) {
        // call kernels 
#ifdef USE_CUSPARSE        
        CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, dBuffer) );
#else
        csr_spmv_adaptable_kernel<ValueType><<<matrixBlockCount, NNZ_PER_WG>>>(device_columnIndices, device_rowStartIndices, device_rowBlocks, device_nnzValues, device_x, device_xSwap); 
#endif
        gather<ValueType><<<gatherGridDim, NNZ_PER_WG>>>(device_b, device_D, device_xSwap, matrixRowCount);

        // Check for convergence
		// Transform: x = abs(x - xSwap)/ xSwap
		thrust::device_ptr<ValueType> devicePtrThrust_x(device_x);
		thrust::device_ptr<ValueType> devicePtrThrust_x_end(device_x + matrixRowCount);
        thrust::device_ptr<ValueType> devicePtrThrust_xSwap(device_xSwap);
        thrust::transform(devicePtrThrust_x, devicePtrThrust_x_end, devicePtrThrust_xSwap, devicePtrThrust_x, equalModuloPrecision<ValueType, Relative>());
        // Reduce: get Max over x and check for res < Precision
		ValueType maxX = thrust::reduce(devicePtrThrust_x, devicePtrThrust_x_end, -std::numeric_limits<ValueType>::max(), thrust::maximum<ValueType>());
		converged = (maxX < precision);
		++iterationCount;

		// Swap pointers, device_x always contains the most current result
        std::swap(device_x, device_xSwap);

#ifdef USE_CUSPARSE
        // set new values for vecX and vecY
        CHECK_CUSPARSE( hipsparseDnVecSetValues(vecX, (void*)device_x) );
        CHECK_CUSPARSE( hipsparseDnVecSetValues(vecY, (void*)device_xSwap) );
#endif
    }

    if (!converged && (iterationCount == maxIterationCount)) {
		iterationCount = 0;
		errorOccured = true;
    }
    
    // Get x (result) back from the device
	CHECK_CUDA( hipMemcpy(x.data(), device_x, sizeof(ValueType) * matrixRowCount, hipMemcpyDeviceToHost) );

    // All code related to freeing memory and clearing up the device
    CHECK_CUDA( hipFree(device_columnIndices) );
    CHECK_CUDA( hipFree(device_rowStartIndices) );
    CHECK_CUDA( hipFree(device_rowBlocks) );
    CHECK_CUDA( hipFree(device_x) );
    CHECK_CUDA( hipFree(device_xSwap) );
    CHECK_CUDA( hipFree(device_b) );
    CHECK_CUDA( hipFree(device_D) );
    CHECK_CUDA( hipFree(device_nnzValues) );

#ifdef USE_CUSPARSE
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) );
    CHECK_CUSPARSE( hipsparseDestroy(handle) );
    CHECK_CUDA( hipFree(dBuffer) );
#endif

    return !errorOccured;
}

template <typename ValueType, bool Maximize, unsigned int THREADS_PER_GROUP>
void __segmented_vector_reduce(const uint_fast64_t num_groups, const uint_fast64_t * nondeterministicChoiceIndices, ValueType * x, const ValueType * y)
{
	const ValueType minMaxInitializer = (Maximize) ? -std::numeric_limits<ValueType>::infinity() : std::numeric_limits<ValueType>::infinity();

    const size_t THREADS_PER_BLOCK  = 128;
    const size_t GROUPS_PER_BLOCK  = THREADS_PER_BLOCK / THREADS_PER_GROUP;
    const size_t NUM_BLOCKS = ceil(double(num_groups) / GROUPS_PER_BLOCK);

    segmented_vector_reduce_kernel<ValueType, GROUPS_PER_BLOCK, THREADS_PER_GROUP, Maximize> <<<NUM_BLOCKS, THREADS_PER_BLOCK>>> (num_groups, nondeterministicChoiceIndices, x, y, minMaxInitializer);
}

template <bool Maximize, typename ValueType>
void segmented_vector_reduce(const uint_fast64_t num_groups, const uint_fast64_t num_entries, const uint_fast64_t * nondeterministicChoiceIndices, ValueType * x, const ValueType * y)
{
    const uint_fast64_t threads_per_group = num_entries / num_groups;

    if (threads_per_group <=  2) { __segmented_vector_reduce<ValueType, Maximize, 2>(num_groups, nondeterministicChoiceIndices, x, y); return; }
    if (threads_per_group <=  4) { __segmented_vector_reduce<ValueType, Maximize, 4>(num_groups, nondeterministicChoiceIndices, x, y); return; }
    if (threads_per_group <=  8) { __segmented_vector_reduce<ValueType, Maximize, 8>(num_groups, nondeterministicChoiceIndices, x, y); return; }
    if (threads_per_group <= 16) { __segmented_vector_reduce<ValueType, Maximize,16>(num_groups, nondeterministicChoiceIndices, x, y); return; }
    
    __segmented_vector_reduce<ValueType, Maximize,32>(num_groups, nondeterministicChoiceIndices, x, y);
}

template <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
bool valueIteration_solver(
                uint_fast64_t const maxIterationCount,
                ValueType const precision, 
                std::vector<uint_fast64_t> const& matrixRowIndices, 
                std::vector<uint_fast64_t> const& columnIndices, 
                std::vector<ValueType> const& nnzValues, 
                std::vector<ValueType> & x, 
                std::vector<ValueType> const& b, 
                std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, 
                size_t& iterationCount, 
                bool const extractScheduler, 
                std::vector<uint_fast64_t>* choices) 
{

    bool errorOccured = false;
    bool converged = false;
    iterationCount = 0;

    uint_fast64_t* device_columnIndices = nullptr;
    uint_fast64_t* device_matrixRowIndices = nullptr;
    uint_fast64_t* device_nondeterministicChoiceIndices = nullptr;

    ValueType* device_nnzValues = nullptr;
    ValueType* device_x = nullptr;
    ValueType* device_xSwap = nullptr;
    ValueType* device_diff = nullptr;
    ValueType* device_b = nullptr;
	ValueType* device_multiplyResult = nullptr;

    const uint_fast64_t matrixRowCount = matrixRowIndices.size() - 1;
    const uint_fast64_t matrixColCount = nondeterministicChoiceIndices.size() - 1;
    const uint_fast64_t matrixNnzCount = nnzValues.size();

    // CUSPARSE APIs
    ValueType            alpha      = 1.0f;
    ValueType            beta       = 0.0f;
    hipsparseHandle_t     handle     = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    
    // CUB APIs
    void     *dTempStorage = NULL;
    size_t   tempStorageBytes = 0;

    // Device memory allocation
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_columnIndices), sizeof(uint_fast64_t) * matrixNnzCount) );
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_matrixRowIndices), sizeof(uint_fast64_t) * (matrixRowCount + 1)) );
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_nondeterministicChoiceIndices), sizeof(uint_fast64_t) * (matrixColCount + 1)) );

    CHECK_CUDA( hipMalloc<ValueType>((&device_nnzValues), sizeof(ValueType) * matrixNnzCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_x), sizeof(ValueType) * matrixColCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_xSwap), sizeof(ValueType) * matrixColCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_diff), sizeof(ValueType) * matrixColCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_b), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_multiplyResult), sizeof(ValueType) * matrixRowCount) );

    // Memory allocated, copy data to device
    CHECK_CUDA( hipMemcpy(device_columnIndices, columnIndices.data(), sizeof(uint_fast64_t) * matrixNnzCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_matrixRowIndices, matrixRowIndices.data(), sizeof(uint_fast64_t) * (matrixRowCount + 1), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_nondeterministicChoiceIndices, nondeterministicChoiceIndices.data(), sizeof(uint_fast64_t) * (matrixColCount + 1), hipMemcpyHostToDevice) );

    CHECK_CUDA( hipMemcpy(device_nnzValues, nnzValues.data(), sizeof(ValueType) * matrixNnzCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_x, x.data(), sizeof(ValueType) * matrixColCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(device_xSwap, 0, sizeof(ValueType) * matrixColCount) );
    CHECK_CUDA( hipMemset(device_diff, 0, sizeof(ValueType) * matrixColCount) );
    CHECK_CUDA( hipMemcpy(device_b, b.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(device_multiplyResult, 0, sizeof(ValueType) * matrixRowCount) );

    // CUSPARSE settings
    CHECK_CUSPARSE( hipsparseCreate(&handle) );
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, matrixRowCount, matrixColCount, matrixNnzCount, (void*)device_matrixRowIndices, (void*)device_columnIndices, (void*)device_nnzValues, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, matrixColCount, (void*)device_x, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, matrixRowCount, (void*)device_multiplyResult, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, &bufferSize) ); 
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) ); 

    // Thrust pointer initialization
    thrust::device_ptr<ValueType> devicePtrThrust_diff(device_diff);
    thrust::device_ptr<ValueType> devicePtrThrust_diff_end(device_diff + matrixColCount);
    thrust::device_ptr<ValueType> devicePtrThrust_b(device_b);
    thrust::device_ptr<ValueType> devicePtrThrust_multiplyResult(device_multiplyResult);

    // Data is on device, start Kernel
    while(!converged && iterationCount < maxIterationCount) {
        /* SPARSE MULT: transition matrix * x vector */
        CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, dBuffer) );

        /* SAXPY: multiplyResult + b inplace to multiplyResult */
		thrust::transform(devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + matrixRowCount, devicePtrThrust_b, devicePtrThrust_multiplyResult, thrust::plus<ValueType>());

        /* MAX/MIN_REDUCE: reduce multiplyResult to a new x vector */
        segmented_vector_reduce<Maximize, ValueType>(matrixColCount, matrixRowCount, device_nondeterministicChoiceIndices, device_xSwap, device_multiplyResult);
        
        /* INF_NORM: check for convergence */
        // Transform: diff = abs(x - xSwap)/ xSwap
		thrust::device_ptr<ValueType> devicePtrThrust_x(device_x);
		thrust::device_ptr<ValueType> devicePtrThrust_x_end(device_x + matrixColCount);
		thrust::device_ptr<ValueType> devicePtrThrust_xSwap(device_xSwap);
		thrust::transform(devicePtrThrust_x, devicePtrThrust_x_end, devicePtrThrust_xSwap, devicePtrThrust_diff, equalModuloPrecision<ValueType, Relative>());
		// Reduce: get Max over x and check for res < Precision
		ValueType maxX = thrust::reduce(devicePtrThrust_diff, devicePtrThrust_diff_end, -std::numeric_limits<ValueType>::max(), thrust::maximum<ValueType>());
		converged = (maxX < precision);
        ++iterationCount;

        // Swap pointers, device_x always contains the most current result
		std::swap(device_x, device_xSwap);
        CHECK_CUSPARSE( hipsparseDnVecSetValues(vecX, (void*)device_x) );
    }

    if (!converged && (iterationCount == maxIterationCount)) {
		iterationCount = 0;
		errorOccured = true;
    }

    // repeat last iteration to extract scheduler
    if (extractScheduler) {
        /* SPARSE MULT: transition matrix * last x before convergence */
        CHECK_CUSPARSE( hipsparseDnVecSetValues(vecX, (void*)device_xSwap) );
        CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, dBuffer) );
        
        /* SAXPY: multiplyResult + b inplace to multiplyResult */
        thrust::transform(devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + matrixRowCount, devicePtrThrust_b, devicePtrThrust_multiplyResult, thrust::plus<ValueType>());
        
        // CUB Memory allocation
        dTempStorage = NULL;
        tempStorageBytes = 0;

        std::vector<hipcub::KeyValuePair<int, ValueType>> host_choices(matrixColCount);
        thrust::device_vector<hipcub::KeyValuePair<int, ValueType>> device_choicesValues(matrixColCount);
        hipcub::KeyValuePair<int, ValueType>* device_choices = thrust::raw_pointer_cast(&device_choicesValues[0]);

        if (Maximize)   hipcub::DeviceSegmentedReduce::ArgMax(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1);
        else            hipcub::DeviceSegmentedReduce::ArgMin(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1); 
        CHECK_CUDA( hipMalloc(&dTempStorage, tempStorageBytes) );
    
        /* MAX/MIN_REDUCE: reduce multiplyResult to a new [(choice,value),...] vector */
        (Maximize) ?
        hipcub::DeviceSegmentedReduce::ArgMax(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1) :
        hipcub::DeviceSegmentedReduce::ArgMin(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1) ; 
    
        /* Copy form device to host and set scheduler choices */
        thrust::copy(device_choicesValues.begin(), device_choicesValues.end(), host_choices.begin());
        for (int i = 0; i < host_choices.size(); i++) {
            choices->at(i) = host_choices[i].key;
        } 
    }

    // Get x (result) back from the device
    CHECK_CUDA( hipMemcpy(x.data(), device_x, sizeof(ValueType) * matrixColCount, hipMemcpyDeviceToHost) );

    // CUSPARSE free
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) );
    CHECK_CUSPARSE( hipsparseDestroy(handle) );

	// All code related to freeing memory and clearing up the device
    CHECK_CUDA( hipFree(device_matrixRowIndices) );
    CHECK_CUDA( hipFree(device_columnIndices) );
    CHECK_CUDA( hipFree(device_nondeterministicChoiceIndices) );
    CHECK_CUDA( hipFree(device_nnzValues) );
    CHECK_CUDA( hipFree(device_x) );
    CHECK_CUDA( hipFree(device_xSwap) );
    CHECK_CUDA( hipFree(device_diff) );
    CHECK_CUDA( hipFree(device_b) );
    CHECK_CUDA( hipFree(device_multiplyResult) );
    CHECK_CUDA( hipFree(dBuffer) );
    CHECK_CUDA( hipFree(dTempStorage) );

    return !errorOccured;
}

template <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
bool valueIteration_solver_multipleMDPs(
                size_t const schedulerSize,
                uint_fast64_t const maxIterationCount,
                ValueType const precision, 
                std::vector<uint_fast64_t> const& matrixRowIndices, 
                std::vector<uint_fast64_t> const& columnIndices, 
                std::vector<ValueType> const& nnzValues, 
                std::vector<ValueType> & x, 
                std::vector<ValueType> const& b, 
                std::vector<uint_fast64_t> const& nondeterministicChoiceIndices,
                std::vector<uint_fast64_t> const& choicesAsKeys,
                size_t& iterationCount, 
                bool const extractScheduler, 
                std::vector<uint_fast64_t>* choices) 
{

    bool errorOccured = false;
    bool converged = false;
    iterationCount = 0;

    uint_fast64_t* device_columnIndices = nullptr;
    uint_fast64_t* device_matrixRowIndices = nullptr;
    uint_fast64_t* device_nondeterministicChoiceIndices = nullptr;

    ValueType* device_nnzValues = nullptr;
    ValueType* device_x = nullptr;
    ValueType* device_xSwap = nullptr;
    ValueType* device_diff = nullptr;
    ValueType* device_b = nullptr;
	ValueType* device_multiplyResult = nullptr;

    const uint_fast64_t matrixRowCount = matrixRowIndices.size() - 1;
    const uint_fast64_t subfamiliesMatrixSize = nondeterministicChoiceIndices.size() - 1;
    const uint_fast64_t familiesCount = b.size() / matrixRowCount;
    const uint_fast64_t matrixColCount = x.size() / familiesCount;
    const uint_fast64_t matrixNnzCount = nnzValues.size();

    // CUSPARSE APIs (matC = matA * matB)
    ValueType            alpha      = 1.0f;
    ValueType            beta       = 0.0f;
    hipsparseHandle_t     handle     = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    // hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    
    // CUB APIs
    void     *dTempStorage = NULL;
    size_t   tempStorageBytes = 0;

    // Device memory allocation
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_columnIndices), sizeof(uint_fast64_t) * matrixNnzCount) );
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_matrixRowIndices), sizeof(uint_fast64_t) * (matrixRowCount + 1)) );
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_nondeterministicChoiceIndices), sizeof(uint_fast64_t) * (subfamiliesMatrixSize + 1)) );

    CHECK_CUDA( hipMalloc<ValueType>((&device_nnzValues), sizeof(ValueType) * matrixNnzCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_x), sizeof(ValueType) * subfamiliesMatrixSize) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_xSwap), sizeof(ValueType) * subfamiliesMatrixSize) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_diff), sizeof(ValueType) * subfamiliesMatrixSize) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_b), sizeof(ValueType) * matrixRowCount * familiesCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_multiplyResult), sizeof(ValueType) * matrixRowCount * familiesCount) );

    // Memory allocated, copy data to device
    CHECK_CUDA( hipMemcpy(device_columnIndices, columnIndices.data(), sizeof(uint_fast64_t) * matrixNnzCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_matrixRowIndices, matrixRowIndices.data(), sizeof(uint_fast64_t) * (matrixRowCount + 1), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_nondeterministicChoiceIndices, nondeterministicChoiceIndices.data(), sizeof(uint_fast64_t) * (subfamiliesMatrixSize + 1), hipMemcpyHostToDevice) );

    CHECK_CUDA( hipMemcpy(device_nnzValues, nnzValues.data(), sizeof(ValueType) * matrixNnzCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_x, x.data(), sizeof(ValueType) * subfamiliesMatrixSize, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(device_xSwap, 0, sizeof(ValueType) * subfamiliesMatrixSize) );
    CHECK_CUDA( hipMemset(device_diff, 0, sizeof(ValueType) * subfamiliesMatrixSize) );
    CHECK_CUDA( hipMemcpy(device_b, b.data(), sizeof(ValueType) * matrixRowCount * familiesCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(device_multiplyResult, 0, sizeof(ValueType) * matrixRowCount * familiesCount) );

    // CUSPARSE settings
    CHECK_CUSPARSE( hipsparseCreate(&handle) );
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, matrixRowCount, matrixColCount, matrixNnzCount, (void*)device_matrixRowIndices, (void*)device_columnIndices, (void*)device_nnzValues, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, matrixColCount, familiesCount, matrixColCount, device_x, CUDA_DATATYPE, HIPSPARSE_ORDER_COL) );
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, matrixRowCount, familiesCount, matrixRowCount, device_multiplyResult, CUDA_DATATYPE, HIPSPARSE_ORDER_COL) );
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, CUDA_DATATYPE, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) ); 
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) ); 

    // Thrust pointer initialization
    thrust::device_ptr<ValueType> devicePtrThrust_diff(device_diff);
    thrust::device_ptr<ValueType> devicePtrThrust_diff_end(device_diff + subfamiliesMatrixSize);
    thrust::device_ptr<ValueType> devicePtrThrust_b(device_b);
    thrust::device_ptr<ValueType> devicePtrThrust_multiplyResult(device_multiplyResult);

    std::vector<ValueType> tmp(matrixRowCount * familiesCount);

    // Data is on device, start Kernel
    while(!converged && iterationCount < maxIterationCount) {
        /* SPARSE MULT: transition matrix * x vector */
        CHECK_CUSPARSE( hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, CUDA_DATATYPE, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) );
        /* SAXPY: multiplyResult + b inplace to multiplyResult */
		thrust::transform(devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + (matrixRowCount * familiesCount), devicePtrThrust_b, devicePtrThrust_multiplyResult, thrust::plus<ValueType>());
        /* MAX/MIN_REDUCE: reduce multiplyResult to a new x vector */
        segmented_vector_reduce<Maximize, ValueType>(subfamiliesMatrixSize, (matrixRowCount * familiesCount), device_nondeterministicChoiceIndices, device_xSwap, device_multiplyResult);
        /* INF_NORM: check for convergence */
        // Transform: diff = abs(x - xSwap)/ xSwap
		thrust::device_ptr<ValueType> devicePtrThrust_x(device_x);
		thrust::device_ptr<ValueType> devicePtrThrust_x_end(device_x + subfamiliesMatrixSize);
		thrust::device_ptr<ValueType> devicePtrThrust_xSwap(device_xSwap);
		thrust::transform(devicePtrThrust_x, devicePtrThrust_x_end, devicePtrThrust_xSwap, devicePtrThrust_diff, equalModuloPrecision<ValueType, Relative>());
		// Reduce: get Max over x and check for res < Precision
		ValueType maxX = thrust::reduce(devicePtrThrust_diff, devicePtrThrust_diff_end, -std::numeric_limits<ValueType>::max(), thrust::maximum<ValueType>());
		converged = (maxX < precision);
        ++iterationCount;

        // Swap pointers, device_x always contains the most current result
		std::swap(device_x, device_xSwap);
        CHECK_CUSPARSE( hipsparseDnMatSetValues(matB, (void*)device_x) );
    }

    if (!converged && (iterationCount == maxIterationCount)) {
		iterationCount = 0;
		errorOccured = true;
    }

    // repeat last iteration to extract scheduler
    if (extractScheduler) {
        /* SPARSE MULT: transition matrix * last x before convergence */
        CHECK_CUSPARSE( hipsparseDnMatSetValues(matB, (void*)device_xSwap) );
        CHECK_CUSPARSE( hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, CUDA_DATATYPE, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) );
        
        /* SAXPY: multiplyResult + b inplace to multiplyResult */
		thrust::transform(devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + (matrixRowCount * familiesCount), devicePtrThrust_b, devicePtrThrust_multiplyResult, thrust::plus<ValueType>());
        
        /* Find scheduler */
        thrust::device_vector<uint_fast64_t> keys(choicesAsKeys);
        // replace +-inf in order to recognize choices to be ignored
        if (Maximize) thrust::replace(thrust::device, devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + (matrixRowCount * familiesCount), -std::numeric_limits<ValueType>::infinity(), ValueType(-1.0));
        else          thrust::replace(thrust::device, devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + (matrixRowCount * familiesCount), std::numeric_limits<ValueType>::infinity(), ValueType(-1.0));
        // remove recognized keys and values
        thrust::device_vector<uint_fast64_t>::iterator newKeysEnd = thrust::remove_if(keys.begin(), keys.end(), devicePtrThrust_multiplyResult, is_minus_one<ValueType>());
        thrust::device_ptr<ValueType> newValuesEnd = thrust::remove(devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + (matrixRowCount * familiesCount), ValueType(-1.0));
        thrust::device_vector<uint_fast64_t> rowGroups(schedulerSize + 1);
        rowGroups[0] = 0;
        // compute new groups without unwanted choices
        thrust::reduce_by_key(keys.begin(), newKeysEnd, thrust::make_constant_iterator<uint_fast64_t>(1), thrust::make_discard_iterator(), rowGroups.begin() + 1, thrust::equal_to<uint_fast64_t>(), thrust::plus<uint_fast64_t>());
        thrust::inclusive_scan(thrust::device, rowGroups.begin() + 1, rowGroups.begin() + schedulerSize + 1, rowGroups.begin() + 1);

        // CUB Memory allocation
        dTempStorage = NULL;
        tempStorageBytes = 0;

        std::vector<hipcub::KeyValuePair<int, ValueType>> host_choices(schedulerSize);
        thrust::device_vector<hipcub::KeyValuePair<int, ValueType>> device_choicesValues(schedulerSize);
        hipcub::KeyValuePair<int, ValueType>* device_choices = thrust::raw_pointer_cast(&device_choicesValues[0]);
        uint_fast64_t *cho = thrust::raw_pointer_cast(&rowGroups[0]);

        if (Maximize)   hipcub::DeviceSegmentedReduce::ArgMax(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, schedulerSize, cho, cho + 1);
        else            hipcub::DeviceSegmentedReduce::ArgMin(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, schedulerSize, cho, cho + 1); 
        CHECK_CUDA( hipMalloc(&dTempStorage, tempStorageBytes) );
    
        /* MAX/MIN_REDUCE: reduce multiplyResult to a new [(choice,value),...] vector */
        (Maximize) ?
        hipcub::DeviceSegmentedReduce::ArgMax(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, schedulerSize, cho, cho + 1) :
        hipcub::DeviceSegmentedReduce::ArgMin(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, schedulerSize, cho, cho + 1) ; 
    
        /* Copy form device to host and set scheduler choices */
        thrust::copy(device_choicesValues.begin(), device_choicesValues.end(), host_choices.begin());
        for (int i = 0; i < host_choices.size(); i++) {
            choices->at(i) = host_choices[i].key;
        } 
    }

    // Get x (result) back from the device
    CHECK_CUDA( hipMemcpy(x.data(), device_x, sizeof(ValueType) * subfamiliesMatrixSize, hipMemcpyDeviceToHost) );

    // CUSPARSE free
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) );
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) );
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) );
    CHECK_CUSPARSE( hipsparseDestroy(handle) );

	// All code related to freeing memory and clearing up the device
    CHECK_CUDA( hipFree(device_matrixRowIndices) );
    CHECK_CUDA( hipFree(device_columnIndices) );
    CHECK_CUDA( hipFree(device_nondeterministicChoiceIndices) );
    CHECK_CUDA( hipFree(device_nnzValues) );
    CHECK_CUDA( hipFree(device_x) );
    CHECK_CUDA( hipFree(device_xSwap) );
    CHECK_CUDA( hipFree(device_diff) );
    CHECK_CUDA( hipFree(device_b) );
    CHECK_CUDA( hipFree(device_multiplyResult) );
    CHECK_CUDA( hipFree(dBuffer) );
    CHECK_CUDA( hipFree(dTempStorage) );

    return !errorOccured;
}

/*******************************************************************************/
/*                    Jacobi Iteration API                                     */
/*******************************************************************************/

bool jacobiIteration_solver_double(uint_fast64_t const maxIterationCount, double const precision, uint_fast64_t const matrixRowCount, uint_fast64_t const matrixNnzCount, uint_fast64_t const matrixBlockCount, std::vector<double> & x, std::vector<double> const& b, std::vector<double> const& nnzValues, std::vector<double> const& D, std::vector<uint_fast64_t> const& columnIndices, std::vector<uint_fast64_t> const& rowStartIndices, std::vector<uint_fast64_t> const& rowBlocks, size_t& iterationCount, bool const relativePrecisionCheck) {
    if (relativePrecisionCheck) {
        return jacobiIteration_solver<true, double, HIP_R_64F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    } else {
        return jacobiIteration_solver<false, double, HIP_R_64F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    }
}

bool jacobiIteration_solver_float(uint_fast64_t const maxIterationCount, float const precision, uint_fast64_t const matrixRowCount, uint_fast64_t const matrixNnzCount, uint_fast64_t const matrixBlockCount, std::vector<float> & x, std::vector<float> const& b, std::vector<float> const& nnzValues, std::vector<float> const& D, std::vector<uint_fast64_t> const& columnIndices, std::vector<uint_fast64_t> const& rowStartIndices, std::vector<uint_fast64_t> const& rowBlocks, size_t& iterationCount, bool const relativePrecisionCheck) {
    if (relativePrecisionCheck) {
        return jacobiIteration_solver<true, float, HIP_R_32F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    } else {
        return jacobiIteration_solver<false, float, HIP_R_32F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    }
}

/*******************************************************************************/
/*                    Value Iteration API                                      */
/*******************************************************************************/

bool valueIteration_solver_uint64_double_minimize(size_t const schedulerSize, bool const solveMultipleInstances, uint_fast64_t const maxIterationCount,double const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<uint_fast64_t> const& columnIndices, std::vector<double> const& nnzValues, std::vector<double>& x, std::vector<double> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, std::vector<uint_fast64_t> const& choicesAsKeys, size_t& iterationCount, bool const extractScheduler, std::vector<uint_fast64_t>* choices) {
    if (relativePrecisionCheck) {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<false, true, double, HIP_R_64F>(schedulerSize, maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, choicesAsKeys, iterationCount, extractScheduler, choices)
               :valueIteration_solver<false, true, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    } else {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<false, false, double, HIP_R_64F>(schedulerSize, maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, choicesAsKeys, iterationCount, extractScheduler, choices)
               :valueIteration_solver<false, false, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    }
}

bool valueIteration_solver_uint64_double_maximize(size_t const schedulerSize, bool const solveMultipleInstances, uint_fast64_t const maxIterationCount,double const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<uint_fast64_t> const& columnIndices, std::vector<double> const& nnzValues, std::vector<double>& x, std::vector<double> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, std::vector<uint_fast64_t> const& choicesAsKeys, size_t& iterationCount, bool const extractScheduler, std::vector<uint_fast64_t>* choices) {
    if (relativePrecisionCheck) {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<true, true, double, HIP_R_64F>(schedulerSize, maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, choicesAsKeys, iterationCount, extractScheduler, choices)
               :valueIteration_solver<true, true, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    } else {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<true, false, double, HIP_R_64F>(schedulerSize, maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, choicesAsKeys, iterationCount, extractScheduler, choices)
               :valueIteration_solver<true, false, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    }
}

bool valueIteration_solver_uint64_float_minimize(size_t const schedulerSize, bool const solveMultipleInstances, uint_fast64_t const maxIterationCount,float const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<uint_fast64_t> const& columnIndices, std::vector<float> const& nnzValues, std::vector<float>& x, std::vector<float> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, std::vector<uint_fast64_t> const& choicesAsKeys, size_t& iterationCount, bool const extractScheduler, std::vector<uint_fast64_t>* choices){
    if (relativePrecisionCheck) {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<false, true, float, HIP_R_32F>(schedulerSize, maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, choicesAsKeys, iterationCount, extractScheduler, choices)
               :valueIteration_solver<false, true, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    } else {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<false, false, float, HIP_R_32F>(schedulerSize, maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, choicesAsKeys, iterationCount, extractScheduler, choices)
               :valueIteration_solver<false, false, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    }
}

bool valueIteration_solver_uint64_float_maximize(size_t const schedulerSize, bool const solveMultipleInstances, uint_fast64_t const maxIterationCount,float const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<uint_fast64_t> const& columnIndices, std::vector<float> const& nnzValues, std::vector<float>& x, std::vector<float> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, std::vector<uint_fast64_t> const& choicesAsKeys, size_t& iterationCount, bool const extractScheduler, std::vector<uint_fast64_t>* choices){
    if (relativePrecisionCheck) {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<true, true, float, HIP_R_32F>(schedulerSize, maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, choicesAsKeys, iterationCount, extractScheduler, choices)
               :valueIteration_solver<true, true, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    } else {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<true, false, float, HIP_R_32F>(schedulerSize, maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, choicesAsKeys, iterationCount, extractScheduler, choices)
               :valueIteration_solver<true, false, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    }
}