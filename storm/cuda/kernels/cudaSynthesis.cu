#include "hip/hip_runtime.h"
#include "cudaSynthesis.h"

#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

#include <cub/device/device_segmented_reduce.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#define FULL_WARP_MASK 0xffffffff
#define USE_CUSPARSE

#define CHECK_CUDA(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA API assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define CHECK_CUSPARSE(func) { cusparseAssert((func), __FILE__, __LINE__); }                                                  
inline void cusparseAssert(hipsparseStatus_t code, const char *file, int line, bool abort=true)
{                                                                             
    if (code != HIPSPARSE_STATUS_SUCCESS) {                                   
        fprintf(stderr,"CUSPARSE API assert: %s %s %d\n", hipsparseGetErrorString(code), file, line);    
        if (abort) exit(code);                                             
    }                                                                          
}

/*******************************************************************************/
/*                             DEVICE CODE                                     */
/*******************************************************************************/

template<typename T, bool Relative>
struct equalModuloPrecision : public thrust::binary_function<T,T,T>
{
__host__ __device__ T operator()(const T &x, const T &y) const
{
    if (Relative) {
		if (y == 0) {
			return ((x >= 0) ? (x) : (-x));
		}
		const T result = (x - y) / y;
		return ((result >= 0) ? (result) : (-result));
    } else {
        const T result = (x - y);
		return ((result >= 0) ? (result) : (-result));
    }
}
};

/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param b 
 * @param D 
 * @param x 
 * @param n 
 */
template<typename ValueType>
__global__ void 
gather(const ValueType *b, const ValueType *D, ValueType *x, const uint_fast64_t n) {
    uint_fast64_t i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < n) {
        x[i] = (b[i] - x[i]) * D[i];
    }
}

/**
 * @brief 
 * 
 * @param x 
 */
__inline__ __device__ 
uint_fast64_t prev_power_of_2 (uint_fast64_t x) {
    
    x = x | (x >> 1);
    x = x | (x >> 2);
    x = x | (x >> 4);
    x = x | (x >> 8);
    x = x | (x >> 16);
    x = x | (x >> 32);
    
    return x - (x >> 1);
}

/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param val 
 */
template<typename ValueType>
__inline__ __device__ 
ValueType warp_reduce (ValueType val) {

    val += __shfl_down_sync(FULL_WARP_MASK, val, 16);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 8);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 4);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 2);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 1);

    return val;
}

/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param col_ids 
 * @param row_ptr 
 * @param row_blocks 
 * @param data 
 * @param x 
 * @param y 
 */
template<typename ValueType>
__global__ void csr_spmv_adaptive_kernel (const uint_fast64_t *col_ids, const uint_fast64_t *row_ptr, const uint_fast64_t *row_blocks, const ValueType *data, const ValueType *x, ValueType *y) {

    const uint_fast64_t block_row_begin = row_blocks[blockIdx.x];
    const uint_fast64_t block_row_end = row_blocks[blockIdx.x + 1];
    const uint_fast64_t nnz_per_block = row_ptr[block_row_end] - row_ptr[block_row_begin]; 

    __shared__ ValueType cache[NNZ_PER_WG];

    if (block_row_end - block_row_begin > 1) {
        // CSR-Stream case...
        const uint_fast64_t i = threadIdx.x;
        const uint_fast64_t block_data_begin = row_ptr[block_row_begin];
        const uint_fast64_t thread_data_begin = block_data_begin + i;

        /**
         * Some block of rows may contain less than NNZ_PER_WG elements. But at 
         * most NNZ_PER_WG elements. (NNZ_PER_WG == blockDim.x -- Each thread has
         * one non-zero element to process.)
         */

        if (i < nnz_per_block) 
            cache[i] = data[thread_data_begin] * x[col_ids[thread_data_begin]];
        __syncthreads ();
        
        /**
         * If matrix is too sparse each thread will reduce 1 row. So one row
         * is reduced by one thread. In case that matrix is not too sparse and there
         * is more non-zero elements in one row than one row is reduced by multiple 
         * threads.
         */
        const uint_fast64_t threads_for_reduction = prev_power_of_2(blockDim.x/(block_row_end - block_row_begin));
            
        if (threads_for_reduction > 1) {
            // Reduce all non zeroes of row by multiple thread
            const uint_fast64_t thread_in_block = i % threads_for_reduction;
            const uint_fast64_t local_row = block_row_begin + i/threads_for_reduction;

            ValueType sum = 0.0;

            if (local_row < block_row_end) {
                const uint_fast64_t local_first_element = row_ptr[local_row] - row_ptr[block_row_begin];
                const uint_fast64_t local_last_element = row_ptr[local_row + 1] - row_ptr[block_row_begin];

                for(uint_fast64_t local_element = local_first_element + thread_in_block;
                    local_element < local_last_element;
                    local_element += threads_for_reduction) 
                    {
                        sum += cache[local_element];    
                    }
            }
            __syncthreads();
            cache[i] = sum;

            // Now each row has "threads_for_reduction" values in cache
            for (int j = threads_for_reduction / 2; j > 0; j >>= 1) {
                // Reduce for each row
                __syncthreads();

                const bool use_result = thread_in_block < j && i + j < NNZ_PER_WG;

                if (use_result)
                    sum += cache[i+j];
                __syncthreads();

                if(use_result)
                    cache[i] = sum;
            }
          
            if(thread_in_block == 0 && local_row < block_row_end)
                y[local_row] = sum;
        
        } else {
            // Reduce all non zeros of row by single thread
            uint_fast64_t local_row = block_row_begin + i;

            while (local_row < block_row_end) {
                ValueType sum = 0.0;

                for (uint_fast64_t j = row_ptr[local_row] - block_data_begin;
                    j < row_ptr[local_row + 1] - block_data_begin;
                    j++) 
                    {
                        sum += cache[j];
                    }
                y[local_row] = sum;
                local_row += NNZ_PER_WG;
            }
        }
    } else {
        /**
         * Row Block == Row => NNZ_PER_WG <= nnz_per_block
         * If a block of rows contains only one row, then it can have 
         * NNZ_PER_WG (blockDim.x) <= nnz_per_block (also blockDim.x > nnz_per_block 
         * is possible). 
         */
        const uint_fast64_t nnz_per_block = row_ptr[block_row_end] - row_ptr[block_row_begin]; 

        if (nnz_per_block <= 64) {
            // CSR-Vector case...
            const uint_fast64_t thread_id = threadIdx.x;
            const uint_fast64_t warp_id = thread_id / warpSize;
            ValueType sum = 0.0;

            if (warp_id == 0) {
                // only one warp processes a whole row
                const uint_fast64_t row_start = row_ptr[block_row_begin];
                const uint_fast64_t row_end = row_ptr[block_row_end];

                for(uint_fast64_t element = row_start + thread_id; element < row_end; element += warpSize) {
                    sum += data[element] * x[col_ids[element]];
                }

                sum = warp_reduce<ValueType>(sum);

                if(thread_id == 0)
                    y[block_row_begin] = sum;
            }
        }  else {
            // CSR-VectorL case...
            const uint_fast64_t thread_id = threadIdx.x;
             
            const uint_fast64_t row_start = row_ptr[block_row_begin];
            const uint_fast64_t row_end = row_ptr[block_row_end];
            
            ValueType sum = 0.0;
            for(uint_fast64_t element = row_start + thread_id; element < row_end; element += blockDim.x)
                sum += data[element] * x[col_ids[element]];

            cache[thread_id] = sum;
            __syncthreads();

            for(int stride = blockDim.x / 2; stride > 0;  stride >>= 1) {
                if(thread_id < stride) 
                    cache[thread_id] += cache[thread_id + stride];
                __syncthreads();
            }

            __syncthreads();

            if(thread_id == 0) 
                y[block_row_begin] = cache[thread_id];
        }
    }
}

/*******************************************************************************/
/*                               HOST CODE                                     */
/*******************************************************************************/

template<bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
bool jacobiIteration_solver(
                uint_fast64_t const maxIterationCount, 
                ValueType const precision,
                uint_fast64_t const matrixRowCount,
                uint_fast64_t const matrixNnzCount,
                uint_fast64_t const matrixBlockCount,
                std::vector<ValueType>& x,
                std::vector<ValueType> const& b,
                std::vector<ValueType> const& nnzValues,
                std::vector<ValueType> const& D,
                std::vector<uint_fast64_t> const& columnIndices,
                std::vector<uint_fast64_t> const& rowStartIndices,
                std::vector<uint_fast64_t> const& rowBlocks,
                size_t& iterationCount)
{
    std::cout << "CUDA Jacobi method\n";

    bool errorOccured = false;
    bool converged = false;
    iterationCount = 0;

    uint_fast64_t* device_columnIndices = nullptr;
    uint_fast64_t* device_rowStartIndices = nullptr; 
    uint_fast64_t* device_rowBlocks = nullptr;

    ValueType* device_x = nullptr;
    ValueType* device_xSwap = nullptr;
    ValueType* device_b = nullptr;
    ValueType* device_D = nullptr;
    ValueType* device_nnzValues = nullptr;

    dim3 gatherGridDim(ceil(double(matrixRowCount)/NNZ_PER_WG));

#ifdef USE_CUSPARSE
    ValueType            alpha      = 1.0f;
    ValueType            beta       = 0.0f;
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
#endif

    // Device memory allocation
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_columnIndices), sizeof(uint_fast64_t) * columnIndices.size()) );
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_rowStartIndices), sizeof(uint_fast64_t) * (matrixRowCount + 1)) );
    
    CHECK_CUDA( hipMalloc<ValueType>((&device_x), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_xSwap), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_b), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_D), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_nnzValues), sizeof(ValueType) * matrixNnzCount) );
#ifndef USE_CUSPARSE
    CHECK_CUDA( cudaMallocResult = hipMalloc<uint_fast64_t>((&device_rowBlocks), sizeof(uint_fast64_t) * (matrixBlockCount + 1)) );
#endif

    // Memory allocated, copy data to device
    CHECK_CUDA( hipMemcpy(device_columnIndices, columnIndices.data(), sizeof(uint_fast64_t) * columnIndices.size(), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_rowStartIndices, rowStartIndices.data(), sizeof(uint_fast64_t) * (matrixRowCount + 1), hipMemcpyHostToDevice) );
    
    CHECK_CUDA( hipMemcpy(device_x, x.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(device_xSwap, 0, sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMemcpy(device_b, b.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_D, D.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_nnzValues, nnzValues.data(), sizeof(ValueType) * matrixNnzCount, hipMemcpyHostToDevice) );
#ifndef USE_CUSPARSE
    CHECK_CUDA( hipMemcpy(device_rowBlocks, rowBlocks.data(), sizeof(uint_fast64_t) * (matrixBlockCount + 1), hipMemcpyHostToDevice) );
#endif

    // CUSPARSE settings
#ifdef USE_CUSPARSE
    CHECK_CUSPARSE( hipsparseCreate(&handle) ); 
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, matrixRowCount, matrixRowCount, matrixNnzCount, (void*)device_rowStartIndices, (void*)device_columnIndices, (void*)device_nnzValues, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, matrixRowCount, (void*)device_x, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, matrixRowCount, (void*)device_xSwap, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, &bufferSize) ); 
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) ); 
#endif

    // Data is on device, start Kernel
    while (!converged && iterationCount < maxIterationCount) {
        // call kernels 
#ifdef USE_CUSPARSE        
        CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, dBuffer) );
#else
        csr_spmv_adaptive_kernel<ValueType><<<matrixBlockCount, NNZ_PER_WG>>>(device_columnIndices, device_rowStartIndices, device_rowBlocks, device_nnzValues, device_x, device_xSwap); 
#endif
        gather<ValueType><<<gatherGridDim, NNZ_PER_WG>>>(device_b, device_D, device_xSwap, matrixRowCount);

        // Check for convergence
		// Transform: x = abs(x - xSwap)/ xSwap
		thrust::device_ptr<ValueType> devicePtrThrust_x(device_x);
		thrust::device_ptr<ValueType> devicePtrThrust_x_end(device_x + matrixRowCount);
        thrust::device_ptr<ValueType> devicePtrThrust_xSwap(device_xSwap);
        thrust::transform(devicePtrThrust_x, devicePtrThrust_x_end, devicePtrThrust_xSwap, devicePtrThrust_x, equalModuloPrecision<ValueType, Relative>());
        // Reduce: get Max over x and check for res < Precision
		ValueType maxX = thrust::reduce(devicePtrThrust_x, devicePtrThrust_x_end, -std::numeric_limits<ValueType>::max(), thrust::maximum<ValueType>());
		converged = (maxX < precision);
		++iterationCount;

		// Swap pointers, device_x always contains the most current result
        std::swap(device_x, device_xSwap);

#ifdef USE_CUSPARSE
        // set new values for vecX and vecY
        CHECK_CUSPARSE( hipsparseDnVecSetValues(vecX, (void*)device_x) );
        CHECK_CUSPARSE( hipsparseDnVecSetValues(vecY, (void*)device_xSwap) );
#endif
    }

    if (!converged && (iterationCount == maxIterationCount)) {
		iterationCount = 0;
		errorOccured = true;
    }
    
    // Get x (result) back from the device
	CHECK_CUDA( hipMemcpy(x.data(), device_x, sizeof(ValueType) * matrixRowCount, hipMemcpyDeviceToHost) );

    // All code related to freeing memory and clearing up the device
    CHECK_CUDA( hipFree(device_columnIndices) );
    CHECK_CUDA( hipFree(device_rowStartIndices) );
    CHECK_CUDA( hipFree(device_rowBlocks) );
    CHECK_CUDA( hipFree(device_x) );
    CHECK_CUDA( hipFree(device_xSwap) );
    CHECK_CUDA( hipFree(device_b) );
    CHECK_CUDA( hipFree(device_D) );
    CHECK_CUDA( hipFree(device_nnzValues) );

#ifdef USE_CUSPARSE
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) );
    CHECK_CUSPARSE( hipsparseDestroy(handle) );
    CHECK_CUDA( hipFree(dBuffer) );
#endif

    std::cout << "--------------------------------------------------------------\n";
    std::cout << "METRICS\n";
    std::cout << "Jacobi Iterations: " << iterationCount << "\n";
    std::cout << "Matrix dimension: " << matrixRowCount << "\n";
    std::cout << "Matrix nnz count: " << matrixNnzCount << "\n";
    std::cout << "--------------------------------------------------------------\n";

    return !errorOccured;
}

template <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
bool valueIteration_solver(
                uint_fast64_t const maxIterationCount,
                ValueType const precision, 
                std::vector<uint_fast64_t> const& matrixRowIndices, 
                std::vector<uint_fast64_t> const& columnIndices, 
                std::vector<ValueType> const& nnzValues, 
                std::vector<ValueType> & x, 
                std::vector<ValueType> const& b, 
                std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, 
                size_t& iterationCount) 
{
    std::cout << "CUDA ValueIteration method\n";

    bool errorOccured = false;
    bool converged = false;
    iterationCount = 0;

    uint_fast64_t* device_columnIndices = nullptr;
    uint_fast64_t* device_matrixRowIndices = nullptr;
    uint_fast64_t* device_nondeterministicChoiceIndices = nullptr;

    ValueType* device_nnzValues = nullptr;
    ValueType* device_x = nullptr;
    ValueType* device_xSwap = nullptr;
    ValueType* device_b = nullptr;
	ValueType* device_multiplyResult = nullptr;

    const uint_fast64_t matrixRowCount = matrixRowIndices.size() - 1;
    const uint_fast64_t matrixColCount = nondeterministicChoiceIndices.size() - 1;
    const uint_fast64_t matrixNnzCount = nnzValues.size();

    // CUSPARSE APIs
    ValueType            alpha      = 1.0f;
    ValueType            beta       = 0.0f;
    hipsparseHandle_t     handle     = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    
    // CUB APIs
    void     *dTempStorage = NULL;
    size_t   tempStorageBytes = 0;

    // Device memory allocation
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_columnIndices), sizeof(uint_fast64_t) * matrixNnzCount) );
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_matrixRowIndices), sizeof(uint_fast64_t) * (matrixRowCount + 1)) );
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_nondeterministicChoiceIndices), sizeof(uint_fast64_t) * (matrixColCount + 1)) );

    CHECK_CUDA( hipMalloc<ValueType>((&device_nnzValues), sizeof(ValueType) * matrixNnzCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_x), sizeof(ValueType) * matrixColCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_xSwap), sizeof(ValueType) * matrixColCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_b), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_multiplyResult), sizeof(ValueType) * matrixRowCount) );

    // Memory allocated, copy data to device
    CHECK_CUDA( hipMemcpy(device_columnIndices, columnIndices.data(), sizeof(uint_fast64_t) * matrixNnzCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_matrixRowIndices, matrixRowIndices.data(), sizeof(uint_fast64_t) * (matrixRowCount + 1), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_nondeterministicChoiceIndices, nondeterministicChoiceIndices.data(), sizeof(uint_fast64_t) * (matrixColCount + 1), hipMemcpyHostToDevice) );

    CHECK_CUDA( hipMemcpy(device_nnzValues, nnzValues.data(), sizeof(ValueType) * matrixNnzCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_x, x.data(), sizeof(ValueType) * matrixColCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(device_xSwap, 0, sizeof(ValueType) * matrixColCount) );
    CHECK_CUDA( hipMemcpy(device_b, b.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(device_multiplyResult, 0, sizeof(ValueType) * matrixRowCount) );

    // CUSPARSE settings
    CHECK_CUSPARSE( hipsparseCreate(&handle) );
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, matrixRowCount, matrixColCount, matrixNnzCount, (void*)device_matrixRowIndices, (void*)device_columnIndices, (void*)device_nnzValues, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, matrixColCount, (void*)device_x, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, matrixRowCount, (void*)device_multiplyResult, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, &bufferSize) ); 
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) ); 

    // CUB Memory allocation
    if (Maximize)   hipcub::DeviceSegmentedReduce::Max(dTempStorage, tempStorageBytes, device_multiplyResult, device_xSwap, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1);
    else            hipcub::DeviceSegmentedReduce::Min(dTempStorage, tempStorageBytes, device_multiplyResult, device_xSwap, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1); 
    CHECK_CUDA( hipMalloc(&dTempStorage, tempStorageBytes) );

    // Data is on device, start Kernel
    while(!converged && iterationCount < maxIterationCount) {
        /* SPARSE MULT: transition matrix * x vector */
        CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, dBuffer) );

        /* SAXPY: multiplyResult + b inplace to multiplyResult */
        thrust::device_ptr<ValueType> devicePtrThrust_b(device_b);
		thrust::device_ptr<ValueType> devicePtrThrust_multiplyResult(device_multiplyResult);
		thrust::transform(devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + matrixRowCount, devicePtrThrust_b, devicePtrThrust_multiplyResult, thrust::plus<ValueType>());

        /* MAX/MIN_REDUCE: reduce multiplyResult to a new x vector */
        (Maximize) ? 
            hipcub::DeviceSegmentedReduce::Max(dTempStorage, tempStorageBytes, device_multiplyResult, device_xSwap, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1) : 
            hipcub::DeviceSegmentedReduce::Min(dTempStorage, tempStorageBytes, device_multiplyResult, device_xSwap, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1) ;

        /* INF_NORM: check for convergence */
        // Transform: x = abs(x - xSwap)/ xSwap
		thrust::device_ptr<ValueType> devicePtrThrust_x(device_x);
		thrust::device_ptr<ValueType> devicePtrThrust_x_end(device_x + matrixColCount);
		thrust::device_ptr<ValueType> devicePtrThrust_xSwap(device_xSwap);
		thrust::transform(devicePtrThrust_x, devicePtrThrust_x_end, devicePtrThrust_xSwap, devicePtrThrust_x, equalModuloPrecision<ValueType, Relative>());
		// Reduce: get Max over x and check for res < Precision
		ValueType maxX = thrust::reduce(devicePtrThrust_x, devicePtrThrust_x_end, -std::numeric_limits<ValueType>::max(), thrust::maximum<ValueType>());
		converged = (maxX < precision);
        ++iterationCount;

        // Swap pointers, device_x always contains the most current result
		std::swap(device_x, device_xSwap);
        CHECK_CUSPARSE( hipsparseDnVecSetValues(vecX, (void*)device_x) );
    }

    if (!converged && (iterationCount == maxIterationCount)) {
		iterationCount = 0;
		errorOccured = true;
    }

    // Get x (result) back from the device
    CHECK_CUDA( hipMemcpy(x.data(), device_x, sizeof(ValueType) * matrixColCount, hipMemcpyDeviceToHost) );

    // CUSPARSE free
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) );
    CHECK_CUSPARSE( hipsparseDestroy(handle) );

	// All code related to freeing memory and clearing up the device
    CHECK_CUDA( hipFree(device_matrixRowIndices) );
    CHECK_CUDA( hipFree(device_columnIndices) );
    CHECK_CUDA( hipFree(device_nondeterministicChoiceIndices) );
    CHECK_CUDA( hipFree(device_nnzValues) );
    CHECK_CUDA( hipFree(device_x) );
    CHECK_CUDA( hipFree(device_xSwap) );
    CHECK_CUDA( hipFree(device_b) );
    CHECK_CUDA( hipFree(device_multiplyResult) );
    CHECK_CUDA( hipFree(dBuffer) );
    CHECK_CUDA( hipFree(dTempStorage) );

    return !errorOccured;
}

/*******************************************************************************/
/*                    Jacobi Iteration API                                     */
/*******************************************************************************/

bool jacobiIteration_solver_double(uint_fast64_t const maxIterationCount, double const precision, uint_fast64_t const matrixRowCount, uint_fast64_t const matrixNnzCount, uint_fast64_t const matrixBlockCount, std::vector<double> & x, std::vector<double> const& b, std::vector<double> const& nnzValues, std::vector<double> const& D, std::vector<uint_fast64_t> const& columnIndices, std::vector<uint_fast64_t> const& rowStartIndices, std::vector<uint_fast64_t> const& rowBlocks, size_t& iterationCount, bool const relativePrecisionCheck) {
    if (relativePrecisionCheck) {
        return jacobiIteration_solver<true, double, HIP_R_64F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    } else {
        return jacobiIteration_solver<false, double, HIP_R_64F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    }
}

bool jacobiIteration_solver_float(uint_fast64_t const maxIterationCount, float const precision, uint_fast64_t const matrixRowCount, uint_fast64_t const matrixNnzCount, uint_fast64_t const matrixBlockCount, std::vector<float> & x, std::vector<float> const& b, std::vector<float> const& nnzValues, std::vector<float> const& D, std::vector<uint_fast64_t> const& columnIndices, std::vector<uint_fast64_t> const& rowStartIndices, std::vector<uint_fast64_t> const& rowBlocks, size_t& iterationCount, bool const relativePrecisionCheck) {
    if (relativePrecisionCheck) {
        return jacobiIteration_solver<true, float, HIP_R_32F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    } else {
        return jacobiIteration_solver<false, float, HIP_R_32F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    }
}

/*******************************************************************************/
/*                    Value Iteration API                                      */
/*******************************************************************************/

bool valueIteration_solver_uint64_double_minimize(uint_fast64_t const maxIterationCount,double const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<uint_fast64_t> const& columnIndices, std::vector<double> const& nnzValues, std::vector<double>& x, std::vector<double> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, size_t& iterationCount) {
    if (relativePrecisionCheck) {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return valueIteration_solver<false, true, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount); 
    } else {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return valueIteration_solver<false, false, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount); 
    }
}

bool valueIteration_solver_uint64_double_maximize(uint_fast64_t const maxIterationCount,double const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<uint_fast64_t> const& columnIndices, std::vector<double> const& nnzValues, std::vector<double>& x, std::vector<double> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, size_t& iterationCount) {
    if (relativePrecisionCheck) {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return valueIteration_solver<true, true, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount); 
    } else {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return valueIteration_solver<true, false, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount); 
    }
}

bool valueIteration_solver_uint64_float_minimize(uint_fast64_t const maxIterationCount,float const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<uint_fast64_t> const& columnIndices, std::vector<float> const& nnzValues, std::vector<float>& x, std::vector<float> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, size_t& iterationCount){
    if (relativePrecisionCheck) {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return valueIteration_solver<false, true, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount); 
    } else {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return valueIteration_solver<false, false, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount); 
    }
}

bool valueIteration_solver_uint64_float_maximize(uint_fast64_t const maxIterationCount,float const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<uint_fast64_t> const& columnIndices, std::vector<float> const& nnzValues, std::vector<float>& x, std::vector<float> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, size_t& iterationCount){
    if (relativePrecisionCheck) {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return valueIteration_solver<true, true, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount); 
    } else {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return valueIteration_solver<true, false, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount); 
    }
}