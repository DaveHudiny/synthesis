#include "hip/hip_runtime.h"
#include "cudaSynthesis.h"

#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>

#include <cub/device/device_segmented_reduce.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#define FULL_WARP_MASK 0xffffffff
#define USE_CUSPARSE

#define CHECK_CUDA(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA API assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define CHECK_CUSPARSE(func) { cusparseAssert((func), __FILE__, __LINE__); }                                                  
inline void cusparseAssert(hipsparseStatus_t code, const char *file, int line, bool abort=true)
{                                                                             
    if (code != HIPSPARSE_STATUS_SUCCESS) {                                   
        fprintf(stderr,"CUSPARSE API assert: %s %s %d\n", hipsparseGetErrorString(code), file, line);    
        if (abort) exit(code);                                             
    }                                                                          
}

/*******************************************************************************/
/*                             DEVICE CODE                                     */
/*******************************************************************************/

template<typename T, bool Relative>
struct equalModuloPrecision : public thrust::binary_function<T,T,T>
{
__host__ __device__ T operator()(const T &x, const T &y) const
{
    if (Relative) {
		if (y == 0) {
			return ((x >= 0) ? (x) : (-x));
		}
		const T result = (x - y) / y;
		return ((result >= 0) ? (result) : (-result));
    } else {
        const T result = (x - y);
		return ((result >= 0) ? (result) : (-result));
    }
}
};

/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param b 
 * @param D 
 * @param x 
 * @param n 
 */
template<typename ValueType>
__global__ void 
gather(const ValueType *b, const ValueType *D, ValueType *x, const uint_fast64_t n) {
    uint_fast64_t i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < n) {
        x[i] = (b[i] - x[i]) * D[i];
    }
}

/**
 * @brief 
 * 
 * @param x 
 */
__inline__ __device__ 
uint_fast64_t prev_power_of_2 (uint_fast64_t x) {
    
    x = x | (x >> 1);
    x = x | (x >> 2);
    x = x | (x >> 4);
    x = x | (x >> 8);
    x = x | (x >> 16);
    x = x | (x >> 32);
    
    return x - (x >> 1);
}

/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param val 
 */
template<typename ValueType>
__inline__ __device__ 
ValueType sumSingleWarpReg (ValueType val) {

    val += __shfl_down_sync(FULL_WARP_MASK, val, 16);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 8);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 4);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 2);
    val += __shfl_down_sync(FULL_WARP_MASK, val, 1);

    return val;
}


/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param val 
 * @param threads_per_row 
 */
 template<typename ValueType>
 __inline__ __device__ 
 ValueType minSingleWarpReg (ValueType val, ValueType minMaxInitializer, uint32_t threads_per_row) {
    ValueType localMin = minMaxInitializer;
    
    switch (threads_per_row) {
        case 32:
            localMin = __shfl_down_sync(FULL_WARP_MASK, val, 16); 
            val = (val < localMin) ? val : localMin;
        case 16:
            localMin = __shfl_down_sync(FULL_WARP_MASK, val, 8); 
            val = (val < localMin) ? val : localMin;
        case 8:
            localMin = __shfl_down_sync(FULL_WARP_MASK, val, 4); 
            val = (val < localMin) ? val : localMin;
        case 4:
            localMin = __shfl_down_sync(FULL_WARP_MASK, val, 2); 
            val = (val < localMin) ? val : localMin;
        case 2:
            localMin = __shfl_down_sync(FULL_WARP_MASK, val, 1); 
            val = (val < localMin) ? val : localMin;
    }

    return val;
 }

 /**
 * @brief 
 * 
 * @tparam ValueType 
 * @param val 
 * @param threads_per_row  
 */
 template<typename ValueType>
 __inline__ __device__ 
 ValueType maxSingleWarpReg (ValueType val, ValueType minMaxInitializer, uint32_t threads_per_row) {
    ValueType localMax = minMaxInitializer;
    
    switch (threads_per_row) {
        case 32:
            localMax = __shfl_down_sync(FULL_WARP_MASK, val, 16); 
            val = (val > localMax) ? val : localMax;
        case 16:
            localMax = __shfl_down_sync(FULL_WARP_MASK, val, 8); 
            val = (val > localMax) ? val : localMax;
        case 8:
            localMax = __shfl_down_sync(FULL_WARP_MASK, val, 4); 
            val = (val > localMax) ? val : localMax;
        case 4:
            localMax = __shfl_down_sync(FULL_WARP_MASK, val, 2); 
            val = (val > localMax) ? val : localMax;
        case 2:
            localMax = __shfl_down_sync(FULL_WARP_MASK, val, 1); 
            val = (val > localMax) ? val : localMax;
    }

    return val;
 }

/**
 * @brief 
 * 
 * @tparam ValueType 
 * @param col_ids 
 * @param row_ptr 
 * @param row_blocks 
 * @param data 
 * @param x 
 * @param y 
 */
template<typename ValueType>
__global__ void csr_spmv_adaptive_kernel (const uint_fast64_t *col_ids, const uint_fast64_t *row_ptr, const uint_fast64_t *row_blocks, const ValueType *data, const ValueType *x, ValueType *y) {

    const uint_fast64_t block_row_begin = row_blocks[blockIdx.x];
    const uint_fast64_t block_row_end = row_blocks[blockIdx.x + 1];
    const uint_fast64_t nnz_per_block = row_ptr[block_row_end] - row_ptr[block_row_begin]; 

    __shared__ ValueType cache[NNZ_PER_WG];

    if (block_row_end - block_row_begin > 1) {
        // CSR-Stream case...
        const uint_fast64_t i = threadIdx.x;
        const uint_fast64_t block_data_begin = row_ptr[block_row_begin];
        const uint_fast64_t thread_data_begin = block_data_begin + i;

        /**
         * Some block of rows may contain less than NNZ_PER_WG elements. But at 
         * most NNZ_PER_WG elements. (NNZ_PER_WG == blockDim.x -- Each thread has
         * one non-zero element to process.)
         */

        if (i < nnz_per_block) 
            cache[i] = data[thread_data_begin] * x[col_ids[thread_data_begin]];
        __syncthreads ();
        
        /**
         * If matrix is too sparse each thread will reduce 1 row. So one row
         * is reduced by one thread. In case that matrix is not too sparse and there
         * is more non-zero elements in one row than one row is reduced by multiple 
         * threads.
         */
        const uint_fast64_t threads_for_reduction = prev_power_of_2(blockDim.x/(block_row_end - block_row_begin));
            
        if (threads_for_reduction > 1) {
            // Reduce all non zeroes of row by multiple thread
            const uint_fast64_t thread_in_block = i % threads_for_reduction;
            const uint_fast64_t local_row = block_row_begin + i/threads_for_reduction;

            ValueType sum = 0.0;

            if (local_row < block_row_end) {
                const uint_fast64_t local_first_element = row_ptr[local_row] - row_ptr[block_row_begin];
                const uint_fast64_t local_last_element = row_ptr[local_row + 1] - row_ptr[block_row_begin];

                for(uint_fast64_t local_element = local_first_element + thread_in_block;
                    local_element < local_last_element;
                    local_element += threads_for_reduction) 
                    {
                        sum += cache[local_element];    
                    }
            }
            __syncthreads();
            cache[i] = sum;

            // Now each row has "threads_for_reduction" values in cache
            for (int j = threads_for_reduction / 2; j > 0; j >>= 1) {
                // Reduce for each row
                __syncthreads();

                const bool use_result = thread_in_block < j && i + j < NNZ_PER_WG;

                if (use_result)
                    sum += cache[i+j];
                __syncthreads();

                if(use_result)
                    cache[i] = sum;
            }
          
            if(thread_in_block == 0 && local_row < block_row_end)
                y[local_row] = sum;
        
        } else {
            // Reduce all non zeros of row by single thread
            uint_fast64_t local_row = block_row_begin + i;

            while (local_row < block_row_end) {
                ValueType sum = 0.0;

                for (uint_fast64_t j = row_ptr[local_row] - block_data_begin;
                    j < row_ptr[local_row + 1] - block_data_begin;
                    j++) 
                    {
                        sum += cache[j];
                    }
                y[local_row] = sum;
                local_row += NNZ_PER_WG;
            }
        }
    } else {
        /**
         * Row Block == Row => NNZ_PER_WG <= nnz_per_block
         * If a block of rows contains only one row, then it can have 
         * NNZ_PER_WG (blockDim.x) <= nnz_per_block (also blockDim.x > nnz_per_block 
         * is possible). 
         */
        const uint_fast64_t nnz_per_block = row_ptr[block_row_end] - row_ptr[block_row_begin]; 

        if (nnz_per_block <= 64) {
            // CSR-Vector case...
            const uint_fast64_t thread_id = threadIdx.x;
            const uint_fast64_t warp_id = thread_id / warpSize;
            ValueType sum = 0.0;

            if (warp_id == 0) {
                // only one warp processes a whole row
                const uint_fast64_t row_start = row_ptr[block_row_begin];
                const uint_fast64_t row_end = row_ptr[block_row_end];

                for(uint_fast64_t element = row_start + thread_id; element < row_end; element += warpSize) {
                    sum += data[element] * x[col_ids[element]];
                }

                sum = sumSingleWarpReg<ValueType>(sum);

                if(thread_id == 0)
                    y[block_row_begin] = sum;
            }
        }  else {
            // CSR-VectorL case...
            const uint_fast64_t thread_id = threadIdx.x;
             
            const uint_fast64_t row_start = row_ptr[block_row_begin];
            const uint_fast64_t row_end = row_ptr[block_row_end];
            
            ValueType sum = 0.0;
            for(uint_fast64_t element = row_start + thread_id; element < row_end; element += blockDim.x)
                sum += data[element] * x[col_ids[element]];

            cache[thread_id] = sum;
            __syncthreads();

            for(int stride = blockDim.x / 2; stride > 0;  stride >>= 1) {
                if(thread_id < stride) 
                    cache[thread_id] += cache[thread_id + stride];
                __syncthreads();
            }

            __syncthreads();

            if(thread_id == 0) 
                y[block_row_begin] = cache[thread_id];
        }
    }
}

template <typename ValueType, unsigned int ROWS_PER_BLOCK, unsigned int THREADS_PER_ROW, bool Maximize>
__launch_bounds__(ROWS_PER_BLOCK * THREADS_PER_ROW,1)
__global__ void
storm_cuda_opt_vector_reduce_kernel(const uint_fast64_t num_groups, const uint_fast64_t * __restrict__ nondeterministicChoiceIndices, ValueType * __restrict__ x, const ValueType * __restrict__ y, const ValueType minMaxInitializer)
{
    __shared__ volatile uint_fast64_t ptrs[ROWS_PER_BLOCK][2];
    
    const uint_fast64_t THREADS_PER_BLOCK = ROWS_PER_BLOCK * THREADS_PER_ROW;

    const uint_fast64_t thread_id   = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x; // global thread index
    const uint_fast64_t thread_lane = threadIdx.x & (THREADS_PER_ROW - 1);          // thread index within the vector
    const uint_fast64_t vector_id   = thread_id   /  THREADS_PER_ROW;               // global vector index
    const uint_fast64_t vector_lane = threadIdx.x /  THREADS_PER_ROW;               // vector index within the block
    const uint_fast64_t num_vectors = ROWS_PER_BLOCK * gridDim.x;                   // total number of active vectors

    for(uint_fast64_t row = vector_id; row < num_groups; row += num_vectors) {
        // use two threads to fetch Ap[row] and Ap[row+1]
        // this is considerably faster than the straightforward version
        if(thread_lane < 2)
            ptrs[vector_lane][thread_lane] = nondeterministicChoiceIndices[row + thread_lane];

        const uint_fast64_t row_start = ptrs[vector_lane][0];                   //same as: row_start = Ap[row];
        const uint_fast64_t row_end   = ptrs[vector_lane][1];                   //same as: row_end   = Ap[row+1];

        // initialize local Min/Max
        ValueType localMinMaxElement = minMaxInitializer;

        if (THREADS_PER_ROW == 32 && row_end - row_start > 32) {
            // ensure aligned memory access to Aj and Ax

            uint_fast64_t jj = row_start - (row_start & (THREADS_PER_ROW - 1)) + thread_lane;

            // accumulate local sums
            if(jj >= row_start && jj < row_end) {
				if(Maximize) { localMinMaxElement = (localMinMaxElement < y[jj]) ? y[jj] : localMinMaxElement; } 
                else         { localMinMaxElement = (localMinMaxElement > y[jj]) ? y[jj] : localMinMaxElement; }
			}

            // accumulate local sums
            for(jj += THREADS_PER_ROW; jj < row_end; jj += THREADS_PER_ROW)
                if(Maximize) { localMinMaxElement = (localMinMaxElement < y[jj]) ? y[jj] : localMinMaxElement; } 
                else         { localMinMaxElement = (localMinMaxElement > y[jj]) ? y[jj] : localMinMaxElement; }
        } else {
            // accumulate local sums
            for(uint_fast64_t jj = row_start + thread_lane; jj < row_end; jj += THREADS_PER_ROW)
                if(Maximize) { localMinMaxElement = (localMinMaxElement < y[jj]) ? y[jj] : localMinMaxElement; } 
                else         { localMinMaxElement = (localMinMaxElement > y[jj]) ? y[jj] : localMinMaxElement; }
        }

        // reduce local min/max to row min/max
        localMinMaxElement = (Maximize) ? maxSingleWarpReg<ValueType>(localMinMaxElement, minMaxInitializer, THREADS_PER_ROW) : 
                                          minSingleWarpReg<ValueType>(localMinMaxElement, minMaxInitializer, THREADS_PER_ROW) ;

        if (thread_lane == 0)
            x[row] = (localMinMaxElement == minMaxInitializer) ? 0.0 : localMinMaxElement;
    }
}

/*******************************************************************************/
/*                               HOST CODE                                     */
/*******************************************************************************/

template<bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
bool jacobiIteration_solver(
                uint_fast64_t const maxIterationCount, 
                ValueType const precision,
                uint_fast64_t const matrixRowCount,
                uint_fast64_t const matrixNnzCount,
                uint_fast64_t const matrixBlockCount,
                std::vector<ValueType>& x,
                std::vector<ValueType> const& b,
                std::vector<ValueType> const& nnzValues,
                std::vector<ValueType> const& D,
                std::vector<uint_fast64_t> const& columnIndices,
                std::vector<uint_fast64_t> const& rowStartIndices,
                std::vector<uint_fast64_t> const& rowBlocks,
                size_t& iterationCount)
{
    std::cout << "CUDA Jacobi method\n";

    bool errorOccured = false;
    bool converged = false;
    iterationCount = 0;

    uint_fast64_t* device_columnIndices = nullptr;
    uint_fast64_t* device_rowStartIndices = nullptr; 
    uint_fast64_t* device_rowBlocks = nullptr;

    ValueType* device_x = nullptr;
    ValueType* device_xSwap = nullptr;
    ValueType* device_b = nullptr;
    ValueType* device_D = nullptr;
    ValueType* device_nnzValues = nullptr;

    dim3 gatherGridDim(ceil(double(matrixRowCount)/NNZ_PER_WG));

#ifdef USE_CUSPARSE
    ValueType            alpha      = 1.0f;
    ValueType            beta       = 0.0f;
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
#endif

    // Device memory allocation
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_columnIndices), sizeof(uint_fast64_t) * columnIndices.size()) );
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_rowStartIndices), sizeof(uint_fast64_t) * (matrixRowCount + 1)) );
    
    CHECK_CUDA( hipMalloc<ValueType>((&device_x), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_xSwap), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_b), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_D), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_nnzValues), sizeof(ValueType) * matrixNnzCount) );
#ifndef USE_CUSPARSE
    CHECK_CUDA( cudaMallocResult = hipMalloc<uint_fast64_t>((&device_rowBlocks), sizeof(uint_fast64_t) * (matrixBlockCount + 1)) );
#endif

    // Memory allocated, copy data to device
    CHECK_CUDA( hipMemcpy(device_columnIndices, columnIndices.data(), sizeof(uint_fast64_t) * columnIndices.size(), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_rowStartIndices, rowStartIndices.data(), sizeof(uint_fast64_t) * (matrixRowCount + 1), hipMemcpyHostToDevice) );
    
    CHECK_CUDA( hipMemcpy(device_x, x.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(device_xSwap, 0, sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMemcpy(device_b, b.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_D, D.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_nnzValues, nnzValues.data(), sizeof(ValueType) * matrixNnzCount, hipMemcpyHostToDevice) );
#ifndef USE_CUSPARSE
    CHECK_CUDA( hipMemcpy(device_rowBlocks, rowBlocks.data(), sizeof(uint_fast64_t) * (matrixBlockCount + 1), hipMemcpyHostToDevice) );
#endif

    // CUSPARSE settings
#ifdef USE_CUSPARSE
    CHECK_CUSPARSE( hipsparseCreate(&handle) ); 
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, matrixRowCount, matrixRowCount, matrixNnzCount, (void*)device_rowStartIndices, (void*)device_columnIndices, (void*)device_nnzValues, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, matrixRowCount, (void*)device_x, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, matrixRowCount, (void*)device_xSwap, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, &bufferSize) ); 
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) ); 
#endif

    // Data is on device, start Kernel
    while (!converged && iterationCount < maxIterationCount) {
        // call kernels 
#ifdef USE_CUSPARSE        
        CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, dBuffer) );
#else
        csr_spmv_adaptive_kernel<ValueType><<<matrixBlockCount, NNZ_PER_WG>>>(device_columnIndices, device_rowStartIndices, device_rowBlocks, device_nnzValues, device_x, device_xSwap); 
#endif
        gather<ValueType><<<gatherGridDim, NNZ_PER_WG>>>(device_b, device_D, device_xSwap, matrixRowCount);

        // Check for convergence
		// Transform: x = abs(x - xSwap)/ xSwap
		thrust::device_ptr<ValueType> devicePtrThrust_x(device_x);
		thrust::device_ptr<ValueType> devicePtrThrust_x_end(device_x + matrixRowCount);
        thrust::device_ptr<ValueType> devicePtrThrust_xSwap(device_xSwap);
        thrust::transform(devicePtrThrust_x, devicePtrThrust_x_end, devicePtrThrust_xSwap, devicePtrThrust_x, equalModuloPrecision<ValueType, Relative>());
        // Reduce: get Max over x and check for res < Precision
		ValueType maxX = thrust::reduce(devicePtrThrust_x, devicePtrThrust_x_end, -std::numeric_limits<ValueType>::max(), thrust::maximum<ValueType>());
		converged = (maxX < precision);
		++iterationCount;

		// Swap pointers, device_x always contains the most current result
        std::swap(device_x, device_xSwap);

#ifdef USE_CUSPARSE
        // set new values for vecX and vecY
        CHECK_CUSPARSE( hipsparseDnVecSetValues(vecX, (void*)device_x) );
        CHECK_CUSPARSE( hipsparseDnVecSetValues(vecY, (void*)device_xSwap) );
#endif
    }

    if (!converged && (iterationCount == maxIterationCount)) {
		iterationCount = 0;
		errorOccured = true;
    }
    
    // Get x (result) back from the device
	CHECK_CUDA( hipMemcpy(x.data(), device_x, sizeof(ValueType) * matrixRowCount, hipMemcpyDeviceToHost) );

    // All code related to freeing memory and clearing up the device
    CHECK_CUDA( hipFree(device_columnIndices) );
    CHECK_CUDA( hipFree(device_rowStartIndices) );
    CHECK_CUDA( hipFree(device_rowBlocks) );
    CHECK_CUDA( hipFree(device_x) );
    CHECK_CUDA( hipFree(device_xSwap) );
    CHECK_CUDA( hipFree(device_b) );
    CHECK_CUDA( hipFree(device_D) );
    CHECK_CUDA( hipFree(device_nnzValues) );

#ifdef USE_CUSPARSE
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) );
    CHECK_CUSPARSE( hipsparseDestroy(handle) );
    CHECK_CUDA( hipFree(dBuffer) );
#endif

    std::cout << "--------------------------------------------------------------\n";
    std::cout << "METRICS\n";
    std::cout << "Jacobi Iterations: " << iterationCount << "\n";
    std::cout << "Matrix dimension: " << matrixRowCount << "\n";
    std::cout << "Matrix nnz count: " << matrixNnzCount << "\n";
    std::cout << "--------------------------------------------------------------\n";

    return !errorOccured;
}

template <typename ValueType, bool Maximize, unsigned int THREADS_PER_VECTOR>
void __storm_cuda_opt_vector_reduce(const uint_fast64_t num_groups, const uint_fast64_t * nondeterministicChoiceIndices, ValueType * x, const ValueType * y)
{
	const ValueType minMaxInitializer = (Maximize) ? -std::numeric_limits<ValueType>::infinity() : std::numeric_limits<ValueType>::infinity();

    const size_t THREADS_PER_BLOCK  = 128;
    const size_t VECTORS_PER_BLOCK  = THREADS_PER_BLOCK / THREADS_PER_VECTOR;

    const size_t NUM_BLOCKS = ceil(double(num_groups) / VECTORS_PER_BLOCK);

    storm_cuda_opt_vector_reduce_kernel<ValueType, VECTORS_PER_BLOCK, THREADS_PER_VECTOR, Maximize> <<<NUM_BLOCKS, THREADS_PER_BLOCK>>> 
        (num_groups, nondeterministicChoiceIndices, x, y, minMaxInitializer);
}

template <bool Maximize, typename ValueType>
void storm_cuda_opt_vector_reduce(const uint_fast64_t num_groups, const uint_fast64_t num_entries, const uint_fast64_t * nondeterministicChoiceIndices, ValueType * x, const ValueType * y)
{
    const uint_fast64_t rows_per_group = num_entries / num_groups;

    if (rows_per_group <=  2) { __storm_cuda_opt_vector_reduce<ValueType, Maximize, 2>(num_groups, nondeterministicChoiceIndices, x, y); return; }
    if (rows_per_group <=  4) { __storm_cuda_opt_vector_reduce<ValueType, Maximize, 4>(num_groups, nondeterministicChoiceIndices, x, y); return; }
    if (rows_per_group <=  8) { __storm_cuda_opt_vector_reduce<ValueType, Maximize, 8>(num_groups, nondeterministicChoiceIndices, x, y); return; }
    if (rows_per_group <= 16) { __storm_cuda_opt_vector_reduce<ValueType, Maximize,16>(num_groups, nondeterministicChoiceIndices, x, y); return; }
    
    __storm_cuda_opt_vector_reduce<ValueType, Maximize,32>(num_groups, nondeterministicChoiceIndices, x, y);
}

template <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
bool valueIteration_solver(
                uint_fast64_t const maxIterationCount,
                ValueType const precision, 
                std::vector<uint_fast64_t> const& matrixRowIndices, 
                std::vector<uint_fast64_t> const& columnIndices, 
                std::vector<ValueType> const& nnzValues, 
                std::vector<ValueType> & x, 
                std::vector<ValueType> const& b, 
                std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, 
                size_t& iterationCount, 
                bool const extractScheduler, 
                std::vector<uint_fast64_t>* choices) 
{
    std::cout << "CUDA ValueIteration method\n";

    bool errorOccured = false;
    bool converged = false;
    iterationCount = 0;

    uint_fast64_t* device_columnIndices = nullptr;
    uint_fast64_t* device_matrixRowIndices = nullptr;
    uint_fast64_t* device_nondeterministicChoiceIndices = nullptr;

    ValueType* device_nnzValues = nullptr;
    ValueType* device_x = nullptr;
    ValueType* device_xSwap = nullptr;
    ValueType* device_diff = nullptr;
    ValueType* device_b = nullptr;
	ValueType* device_multiplyResult = nullptr;

    const uint_fast64_t matrixRowCount = matrixRowIndices.size() - 1;
    const uint_fast64_t matrixColCount = nondeterministicChoiceIndices.size() - 1;
    const uint_fast64_t matrixNnzCount = nnzValues.size();

    // CUSPARSE APIs
    ValueType            alpha      = 1.0f;
    ValueType            beta       = 0.0f;
    hipsparseHandle_t     handle     = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    
    // CUB APIs
    void     *dTempStorage = NULL;
    size_t   tempStorageBytes = 0;

    // Device memory allocation
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_columnIndices), sizeof(uint_fast64_t) * matrixNnzCount) );
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_matrixRowIndices), sizeof(uint_fast64_t) * (matrixRowCount + 1)) );
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_nondeterministicChoiceIndices), sizeof(uint_fast64_t) * (matrixColCount + 1)) );

    CHECK_CUDA( hipMalloc<ValueType>((&device_nnzValues), sizeof(ValueType) * matrixNnzCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_x), sizeof(ValueType) * matrixColCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_xSwap), sizeof(ValueType) * matrixColCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_diff), sizeof(ValueType) * matrixColCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_b), sizeof(ValueType) * matrixRowCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_multiplyResult), sizeof(ValueType) * matrixRowCount) );

    // Memory allocated, copy data to device
    CHECK_CUDA( hipMemcpy(device_columnIndices, columnIndices.data(), sizeof(uint_fast64_t) * matrixNnzCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_matrixRowIndices, matrixRowIndices.data(), sizeof(uint_fast64_t) * (matrixRowCount + 1), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_nondeterministicChoiceIndices, nondeterministicChoiceIndices.data(), sizeof(uint_fast64_t) * (matrixColCount + 1), hipMemcpyHostToDevice) );

    CHECK_CUDA( hipMemcpy(device_nnzValues, nnzValues.data(), sizeof(ValueType) * matrixNnzCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_x, x.data(), sizeof(ValueType) * matrixColCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(device_xSwap, 0, sizeof(ValueType) * matrixColCount) );
    CHECK_CUDA( hipMemset(device_diff, 0, sizeof(ValueType) * matrixColCount) );
    CHECK_CUDA( hipMemcpy(device_b, b.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(device_multiplyResult, 0, sizeof(ValueType) * matrixRowCount) );

    // CUSPARSE settings
    CHECK_CUSPARSE( hipsparseCreate(&handle) );
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, matrixRowCount, matrixColCount, matrixNnzCount, (void*)device_matrixRowIndices, (void*)device_columnIndices, (void*)device_nnzValues, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, matrixColCount, (void*)device_x, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, matrixRowCount, (void*)device_multiplyResult, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, &bufferSize) ); 
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) ); 

    // Thrust pointer initialization
    thrust::device_ptr<ValueType> devicePtrThrust_diff(device_diff);
    thrust::device_ptr<ValueType> devicePtrThrust_diff_end(device_diff + matrixColCount);
    thrust::device_ptr<ValueType> devicePtrThrust_b(device_b);
    thrust::device_ptr<ValueType> devicePtrThrust_multiplyResult(device_multiplyResult);

    // Data is on device, start Kernel
    while(!converged && iterationCount < maxIterationCount) {
        /* SPARSE MULT: transition matrix * x vector */
        CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, dBuffer) );

        /* SAXPY: multiplyResult + b inplace to multiplyResult */
		thrust::transform(devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + matrixRowCount, devicePtrThrust_b, devicePtrThrust_multiplyResult, thrust::plus<ValueType>());

        /* MAX/MIN_REDUCE: reduce multiplyResult to a new x vector */
        storm_cuda_opt_vector_reduce<Maximize, ValueType>(matrixColCount, matrixRowCount, device_nondeterministicChoiceIndices, device_xSwap, device_multiplyResult);
        
        /* INF_NORM: check for convergence */
        // Transform: diff = abs(x - xSwap)/ xSwap
		thrust::device_ptr<ValueType> devicePtrThrust_x(device_x);
		thrust::device_ptr<ValueType> devicePtrThrust_x_end(device_x + matrixColCount);
		thrust::device_ptr<ValueType> devicePtrThrust_xSwap(device_xSwap);
		thrust::transform(devicePtrThrust_x, devicePtrThrust_x_end, devicePtrThrust_xSwap, devicePtrThrust_diff, equalModuloPrecision<ValueType, Relative>());
		// Reduce: get Max over x and check for res < Precision
		ValueType maxX = thrust::reduce(devicePtrThrust_diff, devicePtrThrust_diff_end, -std::numeric_limits<ValueType>::max(), thrust::maximum<ValueType>());
		converged = (maxX < precision);
        ++iterationCount;

        // Swap pointers, device_x always contains the most current result
		std::swap(device_x, device_xSwap);
        CHECK_CUSPARSE( hipsparseDnVecSetValues(vecX, (void*)device_x) );
    }

    if (!converged && (iterationCount == maxIterationCount)) {
		iterationCount = 0;
		errorOccured = true;
    }

    // repeat last iteration to extract scheduler
    if (extractScheduler) {
        /* SPARSE MULT: transition matrix * last x before convergence */
        CHECK_CUDA( hipMemcpy(x.data(), device_xSwap, sizeof(ValueType) * matrixColCount, hipMemcpyDeviceToHost) );

        CHECK_CUSPARSE( hipsparseDnVecSetValues(vecX, (void*)device_xSwap) );
        CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, dBuffer) );
        
        /* SAXPY: multiplyResult + b inplace to multiplyResult */
        thrust::transform(devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + matrixRowCount, devicePtrThrust_b, devicePtrThrust_multiplyResult, thrust::plus<ValueType>());
        
        // CUB Memory allocation
        dTempStorage = NULL;
        tempStorageBytes = 0;

        std::vector<hipcub::KeyValuePair<int, ValueType>> host_choices(matrixColCount);
        thrust::device_vector<hipcub::KeyValuePair<int, ValueType>> device_choicesValues(matrixColCount);
        hipcub::KeyValuePair<int, ValueType>* device_choices = thrust::raw_pointer_cast(&device_choicesValues[0]);

        if (Maximize)   hipcub::DeviceSegmentedReduce::ArgMax(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1);
        else            hipcub::DeviceSegmentedReduce::ArgMin(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1); 
        CHECK_CUDA( hipMalloc(&dTempStorage, tempStorageBytes) );
    
        /* MAX/MIN_REDUCE: reduce multiplyResult to a new [(choice,value),...] vector */
        (Maximize) ?
        hipcub::DeviceSegmentedReduce::ArgMax(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1) :
        hipcub::DeviceSegmentedReduce::ArgMin(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1) ; 
    
        /* Copy form device to host and set scheduler choices */
        thrust::copy(device_choicesValues.begin(), device_choicesValues.end(), host_choices.begin());
        for (int i = 0; i < host_choices.size(); i++) {
            choices->at(i) = host_choices[i].key;
        } 
    }

    // Get x (result) back from the device
    CHECK_CUDA( hipMemcpy(x.data(), device_x, sizeof(ValueType) * matrixColCount, hipMemcpyDeviceToHost) );

    // CUSPARSE free
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) );
    CHECK_CUSPARSE( hipsparseDestroy(handle) );

	// All code related to freeing memory and clearing up the device
    CHECK_CUDA( hipFree(device_matrixRowIndices) );
    CHECK_CUDA( hipFree(device_columnIndices) );
    CHECK_CUDA( hipFree(device_nondeterministicChoiceIndices) );
    CHECK_CUDA( hipFree(device_nnzValues) );
    CHECK_CUDA( hipFree(device_x) );
    CHECK_CUDA( hipFree(device_xSwap) );
    CHECK_CUDA( hipFree(device_diff) );
    CHECK_CUDA( hipFree(device_b) );
    CHECK_CUDA( hipFree(device_multiplyResult) );
    CHECK_CUDA( hipFree(dBuffer) );
    CHECK_CUDA( hipFree(dTempStorage) );

    return !errorOccured;
}

template <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
bool valueIteration_solver_multipleMDPs(
                uint_fast64_t const maxIterationCount,
                ValueType const precision, 
                std::vector<uint_fast64_t> const& matrixRowIndices, 
                std::vector<uint_fast64_t> const& columnIndices, 
                std::vector<ValueType> const& nnzValues, 
                std::vector<ValueType> & x, 
                std::vector<ValueType> const& b, 
                std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, 
                size_t& iterationCount, 
                bool const extractScheduler, 
                std::vector<uint_fast64_t>* choices) 
{
    std::cout << "CUDA ValueIteration method Multiple\n";

    bool errorOccured = false;
    bool converged = false;
    iterationCount = 0;

    uint_fast64_t* device_columnIndices = nullptr;
    uint_fast64_t* device_matrixRowIndices = nullptr;
    uint_fast64_t* device_nondeterministicChoiceIndices = nullptr;

    ValueType* device_nnzValues = nullptr;
    ValueType* device_x = nullptr;
    ValueType* device_xSwap = nullptr;
    ValueType* device_diff = nullptr;
    ValueType* device_b = nullptr;
	ValueType* device_multiplyResult = nullptr;

    const uint_fast64_t matrixRowCount = matrixRowIndices.size() - 1;
    const uint_fast64_t matrixBsizeCount = nondeterministicChoiceIndices.size() - 1;
    const uint_fast64_t familiesCount = b.size() / matrixRowCount;
    const uint_fast64_t matrixColCount = x.size() / familiesCount;
    const uint_fast64_t matrixNnzCount = nnzValues.size();

    // CUSPARSE APIs (matC = matA * matB)
    ValueType            alpha      = 1.0f;
    ValueType            beta       = 0.0f;
    hipsparseHandle_t     handle     = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    // hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    
    // CUB APIs
    void     *dTempStorage = NULL;
    size_t   tempStorageBytes = 0;

    // memory needed
    // sizeof(uint_fast64_t) * matrixNnzCount
    // sizeof(uint_fast64_t) * matrixRowCount + 1
    // sizeof(uint_fast64_t) * matrixBsizeCount + 1
    // sizeof(ValueType)     * matrixNnzCount
    // sizeof(ValueType)     * matrixBsizeCount
    // sizeof(ValueType)     * matrixBsizeCount
    // sizeof(ValueType)     * matrixBsizeCount
    // sizeof(ValueType)     * matrixRowCount * familiesCount
    // sizeof(ValueType)     * matrixRowCount * familiesCount


    // Device memory allocation
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_columnIndices), sizeof(uint_fast64_t) * matrixNnzCount) );
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_matrixRowIndices), sizeof(uint_fast64_t) * (matrixRowCount + 1)) );
    CHECK_CUDA( hipMalloc<uint_fast64_t>((&device_nondeterministicChoiceIndices), sizeof(uint_fast64_t) * (matrixBsizeCount + 1)) );

    CHECK_CUDA( hipMalloc<ValueType>((&device_nnzValues), sizeof(ValueType) * matrixNnzCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_x), sizeof(ValueType) * matrixBsizeCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_xSwap), sizeof(ValueType) * matrixBsizeCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_diff), sizeof(ValueType) * matrixBsizeCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_b), sizeof(ValueType) * matrixRowCount * familiesCount) );
    CHECK_CUDA( hipMalloc<ValueType>((&device_multiplyResult), sizeof(ValueType) * matrixRowCount * familiesCount) );

    // Memory allocated, copy data to device
    CHECK_CUDA( hipMemcpy(device_columnIndices, columnIndices.data(), sizeof(uint_fast64_t) * matrixNnzCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_matrixRowIndices, matrixRowIndices.data(), sizeof(uint_fast64_t) * (matrixRowCount + 1), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_nondeterministicChoiceIndices, nondeterministicChoiceIndices.data(), sizeof(uint_fast64_t) * (matrixBsizeCount + 1), hipMemcpyHostToDevice) );

    CHECK_CUDA( hipMemcpy(device_nnzValues, nnzValues.data(), sizeof(ValueType) * matrixNnzCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(device_x, x.data(), sizeof(ValueType) * matrixBsizeCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(device_xSwap, 0, sizeof(ValueType) * matrixBsizeCount) );
    CHECK_CUDA( hipMemset(device_diff, 0, sizeof(ValueType) * matrixBsizeCount) );
    CHECK_CUDA( hipMemcpy(device_b, b.data(), sizeof(ValueType) * matrixRowCount * familiesCount, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(device_multiplyResult, 0, sizeof(ValueType) * matrixRowCount * familiesCount) );

    // CUSPARSE settings
    CHECK_CUSPARSE( hipsparseCreate(&handle) );
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, matrixRowCount, matrixColCount, matrixNnzCount, (void*)device_matrixRowIndices, (void*)device_columnIndices, (void*)device_nnzValues, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, CUDA_DATATYPE) );
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, matrixColCount, familiesCount, matrixColCount, device_x, CUDA_DATATYPE, HIPSPARSE_ORDER_COL) );
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, matrixRowCount, familiesCount, matrixRowCount, device_multiplyResult, CUDA_DATATYPE, HIPSPARSE_ORDER_COL) );
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, CUDA_DATATYPE, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) ); 
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) ); 

    // Thrust pointer initialization
    thrust::device_ptr<ValueType> devicePtrThrust_diff(device_diff);
    thrust::device_ptr<ValueType> devicePtrThrust_diff_end(device_diff + matrixColCount);
    thrust::device_ptr<ValueType> devicePtrThrust_b(device_b);
    thrust::device_ptr<ValueType> devicePtrThrust_multiplyResult(device_multiplyResult);

    std::vector<ValueType> tmp(matrixRowCount * familiesCount);

    // Data is on device, start Kernel
    while(!converged && iterationCount < maxIterationCount) {
        /* SPARSE MULT: transition matrix * x vector */
        // CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, dBuffer) );
        CHECK_CUSPARSE( hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, CUDA_DATATYPE, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) );
        
        hipMemcpy(tmp.data(), device_multiplyResult, sizeof(ValueType) * matrixRowCount * familiesCount, hipMemcpyDeviceToHost);
        std::cout << "mult: ";
        for(int i = 0; i< tmp.size(); i++) {
            std::cout << tmp.at(i) << " ";
        } std::cout << "\n";
        /* SAXPY: multiplyResult + b inplace to multiplyResult */
		thrust::transform(devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + (matrixRowCount * familiesCount), devicePtrThrust_b, devicePtrThrust_multiplyResult, thrust::plus<ValueType>());
        hipMemcpy(tmp.data(), device_multiplyResult, sizeof(ValueType) * matrixRowCount * familiesCount, hipMemcpyDeviceToHost);
        std::cout << "add : ";
        for(int i = 0; i< tmp.size(); i++) {
            std::cout << tmp.at(i) << " ";
        } std::cout << "\n";

        /* MAX/MIN_REDUCE: reduce multiplyResult to a new x vector */
        storm_cuda_opt_vector_reduce<Maximize, ValueType>(matrixBsizeCount, (matrixRowCount * familiesCount), device_nondeterministicChoiceIndices, device_xSwap, device_multiplyResult);
        hipMemcpy(x.data(), device_xSwap, sizeof(ValueType) * matrixBsizeCount, hipMemcpyDeviceToHost);
        std::cout << "redu: ";
        for(int i = 0; i< x.size(); i++) {
            std::cout << x.at(i) << " ";
        } std::cout << "\n";

        /* INF_NORM: check for convergence */
        // Transform: diff = abs(x - xSwap)/ xSwap
		thrust::device_ptr<ValueType> devicePtrThrust_x(device_x);
		thrust::device_ptr<ValueType> devicePtrThrust_x_end(device_x + matrixColCount);
		thrust::device_ptr<ValueType> devicePtrThrust_xSwap(device_xSwap);
		thrust::transform(devicePtrThrust_x, devicePtrThrust_x_end, devicePtrThrust_xSwap, devicePtrThrust_diff, equalModuloPrecision<ValueType, Relative>());
		// Reduce: get Max over x and check for res < Precision
		ValueType maxX = thrust::reduce(devicePtrThrust_diff, devicePtrThrust_diff_end, -std::numeric_limits<ValueType>::max(), thrust::maximum<ValueType>());
		converged = (maxX < precision);
        ++iterationCount;

        // Swap pointers, device_x always contains the most current result
		std::swap(device_x, device_xSwap);
        CHECK_CUSPARSE( hipsparseDnMatSetValues(matB, (void*)device_x) );
    }

    if (!converged && (iterationCount == maxIterationCount)) {
		iterationCount = 0;
		errorOccured = true;
    }

    // repeat last iteration to extract scheduler
    // if (extractScheduler) {
    //     /* SPARSE MULT: transition matrix * last x before convergence */
    //     CHECK_CUDA( hipMemcpy(x.data(), device_xSwap, sizeof(ValueType) * matrixColCount, hipMemcpyDeviceToHost) );

    //     CHECK_CUSPARSE( hipsparseDnVecSetValues(vecX, (void*)device_xSwap) );
    //     CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, CUDA_DATATYPE, HIPSPARSE_CSRMV_ALG2, dBuffer) );
        
    //     /* SAXPY: multiplyResult + b inplace to multiplyResult */
    //     thrust::transform(devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + matrixRowCount, devicePtrThrust_b, devicePtrThrust_multiplyResult, thrust::plus<ValueType>());
        
    //     // CUB Memory allocation
    //     dTempStorage = NULL;
    //     tempStorageBytes = 0;

    //     std::vector<hipcub::KeyValuePair<int, ValueType>> host_choices(matrixColCount);
    //     thrust::device_vector<hipcub::KeyValuePair<int, ValueType>> device_choicesValues(matrixColCount);
    //     hipcub::KeyValuePair<int, ValueType>* device_choices = thrust::raw_pointer_cast(&device_choicesValues[0]);

    //     if (Maximize)   hipcub::DeviceSegmentedReduce::ArgMax(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1);
    //     else            hipcub::DeviceSegmentedReduce::ArgMin(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1); 
    //     CHECK_CUDA( hipMalloc(&dTempStorage, tempStorageBytes) );
    
    //     /* MAX/MIN_REDUCE: reduce multiplyResult to a new [(choice,value),...] vector */
    //     (Maximize) ?
    //     hipcub::DeviceSegmentedReduce::ArgMax(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1) :
    //     hipcub::DeviceSegmentedReduce::ArgMin(dTempStorage, tempStorageBytes, device_multiplyResult, device_choices, matrixColCount, device_nondeterministicChoiceIndices, device_nondeterministicChoiceIndices + 1) ; 
    
    //     /* Copy form device to host and set scheduler choices */
    //     thrust::copy(device_choicesValues.begin(), device_choicesValues.end(), host_choices.begin());
    //     for (int i = 0; i < host_choices.size(); i++) {
    //         choices->at(i) = host_choices[i].key;
    //     } 
    // }

    // Get x (result) back from the device
    CHECK_CUDA( hipMemcpy(x.data(), device_x, sizeof(ValueType) * matrixBsizeCount, hipMemcpyDeviceToHost) );

    // CUSPARSE free
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) );
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) );
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) );
    CHECK_CUSPARSE( hipsparseDestroy(handle) );

	// All code related to freeing memory and clearing up the device
    CHECK_CUDA( hipFree(device_matrixRowIndices) );
    CHECK_CUDA( hipFree(device_columnIndices) );
    CHECK_CUDA( hipFree(device_nondeterministicChoiceIndices) );
    CHECK_CUDA( hipFree(device_nnzValues) );
    CHECK_CUDA( hipFree(device_x) );
    CHECK_CUDA( hipFree(device_xSwap) );
    CHECK_CUDA( hipFree(device_diff) );
    CHECK_CUDA( hipFree(device_b) );
    CHECK_CUDA( hipFree(device_multiplyResult) );
    CHECK_CUDA( hipFree(dBuffer) );
    CHECK_CUDA( hipFree(dTempStorage) );

    return !errorOccured;
}

/*******************************************************************************/
/*                    Jacobi Iteration API                                     */
/*******************************************************************************/

bool jacobiIteration_solver_double(uint_fast64_t const maxIterationCount, double const precision, uint_fast64_t const matrixRowCount, uint_fast64_t const matrixNnzCount, uint_fast64_t const matrixBlockCount, std::vector<double> & x, std::vector<double> const& b, std::vector<double> const& nnzValues, std::vector<double> const& D, std::vector<uint_fast64_t> const& columnIndices, std::vector<uint_fast64_t> const& rowStartIndices, std::vector<uint_fast64_t> const& rowBlocks, size_t& iterationCount, bool const relativePrecisionCheck) {
    if (relativePrecisionCheck) {
        return jacobiIteration_solver<true, double, HIP_R_64F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    } else {
        return jacobiIteration_solver<false, double, HIP_R_64F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    }
}

bool jacobiIteration_solver_float(uint_fast64_t const maxIterationCount, float const precision, uint_fast64_t const matrixRowCount, uint_fast64_t const matrixNnzCount, uint_fast64_t const matrixBlockCount, std::vector<float> & x, std::vector<float> const& b, std::vector<float> const& nnzValues, std::vector<float> const& D, std::vector<uint_fast64_t> const& columnIndices, std::vector<uint_fast64_t> const& rowStartIndices, std::vector<uint_fast64_t> const& rowBlocks, size_t& iterationCount, bool const relativePrecisionCheck) {
    if (relativePrecisionCheck) {
        return jacobiIteration_solver<true, float, HIP_R_32F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    } else {
        return jacobiIteration_solver<false, float, HIP_R_32F>(maxIterationCount, precision, matrixRowCount, matrixNnzCount, matrixBlockCount, x, b, nnzValues, D, columnIndices, rowStartIndices, rowBlocks, iterationCount);
    }
}

/*******************************************************************************/
/*                    Value Iteration API                                      */
/*******************************************************************************/

bool valueIteration_solver_uint64_double_minimize(bool const solveMultipleInstances, uint_fast64_t const maxIterationCount,double const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<uint_fast64_t> const& columnIndices, std::vector<double> const& nnzValues, std::vector<double>& x, std::vector<double> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, size_t& iterationCount, bool const extractScheduler, std::vector<uint_fast64_t>* choices) {
    if (relativePrecisionCheck) {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<false, true, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices)
               :valueIteration_solver<false, true, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    } else {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<false, false, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices)
               :valueIteration_solver<false, false, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    }
}

bool valueIteration_solver_uint64_double_maximize(bool const solveMultipleInstances, uint_fast64_t const maxIterationCount,double const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<uint_fast64_t> const& columnIndices, std::vector<double> const& nnzValues, std::vector<double>& x, std::vector<double> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, size_t& iterationCount, bool const extractScheduler, std::vector<uint_fast64_t>* choices) {
    if (relativePrecisionCheck) {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<true, true, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices)
               :valueIteration_solver<true, true, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    } else {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<true, false, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices)
               :valueIteration_solver<true, false, double, HIP_R_64F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    }
}

bool valueIteration_solver_uint64_float_minimize(bool const solveMultipleInstances, uint_fast64_t const maxIterationCount,float const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<uint_fast64_t> const& columnIndices, std::vector<float> const& nnzValues, std::vector<float>& x, std::vector<float> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, size_t& iterationCount, bool const extractScheduler, std::vector<uint_fast64_t>* choices){
    if (relativePrecisionCheck) {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<false, true, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices)
               :valueIteration_solver<false, true, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    } else {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<false, false, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices)
               :valueIteration_solver<false, false, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    }
}

bool valueIteration_solver_uint64_float_maximize(bool const solveMultipleInstances, uint_fast64_t const maxIterationCount,float const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<uint_fast64_t> const& columnIndices, std::vector<float> const& nnzValues, std::vector<float>& x, std::vector<float> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, size_t& iterationCount, bool const extractScheduler, std::vector<uint_fast64_t>* choices){
    if (relativePrecisionCheck) {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<true, true, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices)
               :valueIteration_solver<true, true, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    } else {
        // <bool Maximize, bool Relative, typename ValueType, hipDataType CUDA_DATATYPE>
        return (solveMultipleInstances) ?
                valueIteration_solver_multipleMDPs<true, false, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices)
               :valueIteration_solver<true, false, float, HIP_R_32F>(maxIterationCount, precision, matrixRowIndices, columnIndices, nnzValues, x, b, nondeterministicChoiceIndices, iterationCount, extractScheduler, choices);  
    }
}